#include "hip/hip_runtime.h"
//HEAD_DSPH
/*
<DUALSPHYSICS>  Copyright (c) 2019 by Dr Jose M. Dominguez et al. (see http://dual.sphysics.org/index.php/developers/).

EPHYSLAB Environmental Physics Laboratory, Universidade de Vigo, Ourense, Spain.
School of Mechanical, Aerospace and Civil Engineering, University of Manchester, Manchester, U.K.

This file is part of DualSPHysics.

DualSPHysics is free software: you can redistribute it and/or modify it under the terms of the GNU Lesser General Public License
as published by the Free Software Foundation; either version 2.1 of the License, or (at your option) any later version.

DualSPHysics is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more details.

You should have received a copy of the GNU Lesser General Public License along with DualSPHysics. If not, see <http://www.gnu.org/licenses/>.
*/

/// \file JSphGpu_ker.cu \brief Implements functions and CUDA kernels for the Particle Interaction and System Update.

#include "JSphGpu_NN_ker.h"
//#include <cfloat>
//#include <hip/hip_math_constants.h>

#define MAXNUMBERPHASE 10

__constant__ StPhaseCte PHASECTE[MAXNUMBERPHASE];
__constant__ StPhaseArray PHASEARRAY[MAXNUMBERPHASE];
// __constant__ StPhaseHypo PHASEHYPO[MAXNUMBERPHASE];
// __constant__ StPhaseElastic PHASEELASTIC[MAXNUMBERPHASE];

namespace cusphNN {
#include "FunctionsBasic_iker.h" //orig
#include "FunctionsMath_iker.h" //orig
#include "FunSphKernel_iker.h"
#include "FunSphEos_iker.h"
#undef _JCellSearch_iker_
#include "JCellSearch_iker.h"


//==============================================================================
/// Stores constants for the GPU interaction.
/// Graba constantes para la interaccion a la GPU.
//==============================================================================
void CteInteractionUp_NN(unsigned phasecount,const StPhaseCte *phasecte,const StPhaseArray *phasearray){
  hipMemcpyToSymbol(HIP_SYMBOL(PHASECTE),phasecte,sizeof(StPhaseCte)*phasecount);
  hipMemcpyToSymbol(HIP_SYMBOL(PHASEARRAY),phasearray,sizeof(StPhaseArray)*phasecount);
}
void CteInteractionUp_NN(unsigned phasecount, const StPhaseHypo *phasehypo){
  hipMemcpyToSymbol(HIP_SYMBOL(PHASEHYPO),phasehypo,sizeof(StPhaseHypo)*phasecount);
}

void CteInteractionUp_NN(unsigned phasecount, const StPhaseElastic *phaseelastic){
  hipMemcpyToSymbol(HIP_SYMBOL(PHASEELASTIC),phaseelastic,sizeof(StPhaseElastic)*phasecount);
}
//------------------------------------------------------------------------------
/// Doubles the position of the indicated particle using a displacement.
/// Duplicate particles are considered valid and are always within
/// the domain.
/// This kernel applies to single-GPU and multi-GPU because the calculations are made
/// from domposmin.
/// It controls the cell coordinates not exceed the maximum.
///
/// Duplica la posicion de la particula indicada aplicandole un desplazamiento.
/// Las particulas duplicadas se considera que siempre son validas y estan dentro
/// del dominio.
/// Este kernel vale para single-gpu y multi-gpu porque los calculos se hacen 
/// a partir de domposmin.
/// Se controla que las coordendas de celda no sobrepasen el maximo.
//------------------------------------------------------------------------------
__device__ void KerPeriodicDuplicatePos(unsigned pnew,unsigned pcopy
  ,bool inverse,double dx,double dy,double dz,uint3 cellmax
  ,double2 *posxy,double *posz,unsigned *dcell)
{
  //-Obtains position of the particle to be duplicated.
  //-Obtiene pos de particula a duplicar.
  double2 rxy=posxy[pcopy];
  double rz=posz[pcopy];
  //-Applies displacement.
  rxy.x+=(inverse ? -dx : dx);
  rxy.y+=(inverse ? -dy : dy);
  rz+=(inverse ? -dz : dz);
  //-Computes cell coordinates within the domain.
  //-Calcula coordendas de celda dentro de dominio.
  unsigned cx=unsigned((rxy.x-CTE.domposminx)/CTE.scell);
  unsigned cy=unsigned((rxy.y-CTE.domposminy)/CTE.scell);
  unsigned cz=unsigned((rz-CTE.domposminz)/CTE.scell);
  //-Adjust cell coordinates if they exceed the maximum.
  //-Ajusta las coordendas de celda si sobrepasan el maximo.
  cx=(cx<=cellmax.x ? cx : cellmax.x);
  cy=(cy<=cellmax.y ? cy : cellmax.y);
  cz=(cz<=cellmax.z ? cz : cellmax.z);
  //-Stores position and cell of the new particles.
  //-Graba posicion y celda de nuevas particulas.
  posxy[pnew]=rxy;
  posz[pnew]=rz;
  dcell[pnew]=PC__Cell(CTE.cellcode,cx,cy,cz);
}
//------------------------------------------------------------------------------
/// Creates periodic particles from a list of particles to duplicate for non-Newtonian models.
/// It is assumed that all particles are valid.
/// This kernel applies to single-GPU and multi-GPU because it uses domposmin.
///
/// Crea particulas periodicas a partir de una lista con las particulas a duplicar.
/// Se presupone que todas las particulas son validas.
/// Este kernel vale para single-gpu y multi-gpu porque usa domposmin. 
//------------------------------------------------------------------------------
template<bool varspre> __global__ void KerPeriodicDuplicateSymplectic_NN(unsigned n,unsigned pini
  ,uint3 cellmax,double3 perinc,const unsigned *listp,unsigned *idp,typecode *code,unsigned *dcell
  ,double2 *posxy,double *posz,float4 *velrhop,float *auxnn,double2 *posxypre,double *poszpre,float4 *velrhoppre)
{
  const unsigned p=blockIdx.x*blockDim.x+threadIdx.x; //-Number of particle.
  if(p<n) {
    const unsigned pnew=p+pini;
    const unsigned rp=listp[p];
    const unsigned pcopy=(rp&0x7FFFFFFF);
    //-Adjusts cell position of the new particles.
    //-Ajusta posicion y celda de nueva particula.
    KerPeriodicDuplicatePos(pnew,pcopy,(rp>=0x80000000),perinc.x,perinc.y,perinc.z,cellmax,posxy,posz,dcell);
    //-Copies the remaining data.
    //-Copia el resto de datos.
    idp[pnew]=idp[pcopy];
    code[pnew]=CODE_SetPeriodic(code[pcopy]);
    velrhop[pnew]=velrhop[pcopy];
    if(varspre) {
      posxypre[pnew]=posxypre[pcopy];
      poszpre[pnew]=poszpre[pcopy];
      velrhoppre[pnew]=velrhoppre[pcopy];
    }
    if(auxnn)auxnn[pnew]=auxnn[pcopy];
  }
}

//==============================================================================
/// Creates periodic particles from a list of particles to duplicate for non-Newotnian formulation..
/// Crea particulas periodicas a partir de una lista con las particulas a duplicar.
//==============================================================================
void PeriodicDuplicateSymplectic(unsigned n,unsigned pini
  ,tuint3 domcells,tdouble3 perinc,const unsigned *listp,unsigned *idp,typecode *code,unsigned *dcell
  ,double2 *posxy,double *posz,float4 *velrhop,float *auxnn,double2 *posxypre,double *poszpre,float4 *velrhoppre)
{
  if(n) {
    uint3 cellmax=make_uint3(domcells.x-1,domcells.y-1,domcells.z-1);
    dim3 sgrid=GetSimpleGridSize(n,SPHBSIZE);
    if(posxypre!=NULL)KerPeriodicDuplicateSymplectic_NN<true><<<sgrid,SPHBSIZE>>>(n,pini,cellmax,Double3(perinc),listp,idp,code,dcell,posxy,posz,velrhop,auxnn,posxypre,poszpre,velrhoppre);
    else                 KerPeriodicDuplicateSymplectic_NN<false><<<sgrid,SPHBSIZE>>>(n,pini,cellmax,Double3(perinc),listp,idp,code,dcell,posxy,posz,velrhop,auxnn,posxypre,poszpre,velrhoppre);
  }
}

//------------------------------------------------------------------------------
/// Creates periodic particles from a list of particles to duplicate.
/// It is assumed that all particles are valid.
/// This kernel applies to single-GPU and multi-GPU because it uses domposmin.
///
/// Crea particulas periodicas a partir de una lista con las particulas a duplicar.
/// Se presupone que todas las particulas son validas.
/// Este kernel vale para single-gpu y multi-gpu porque usa domposmin. 
//------------------------------------------------------------------------------
/*
__global__ void KerPeriodicDuplicateVerlet(unsigned n,unsigned pini,uint3 cellmax,double3 perinc
  ,const unsigned *listp,unsigned *idp,typecode *code,unsigned *dcell
  ,double2 *posxy,double *posz,float4 *velrhop,float *auxnn,float4 *velrhopm1)
{
  const unsigned p=blockIdx.x*blockDim.x+threadIdx.x; //-Number of particle.
  if(p<n) {
    const unsigned pnew=p+pini;
    const unsigned rp=listp[p];
    const unsigned pcopy=(rp&0x7FFFFFFF);
    //-Adjusts cell position of the new particles.
    //-Ajusta posicion y celda de nueva particula.
    KerPeriodicDuplicatePos(pnew,pcopy,(rp>=0x80000000),perinc.x,perinc.y,perinc.z,cellmax,posxy,posz,dcell);
    //-Copies the remaining data.
    //-Copia el resto de datos.
    idp[pnew]=idp[pcopy];
    code[pnew]=CODE_SetPeriodic(code[pcopy]);
    velrhop[pnew]=velrhop[pcopy];
    velrhopm1[pnew]=velrhopm1[pcopy];
    if(auxnn)auxnn[pnew]=auxnn[pcopy];
  }
}
*/
//==============================================================================
/// Creates periodic particles from a list of particles to duplicate.
/// Crea particulas periodicas a partir de una lista con las particulas a duplicar.
//==============================================================================
/*
void PeriodicDuplicateVerlet(unsigned n,unsigned pini,tuint3 domcells,tdouble3 perinc
  ,const unsigned *listp,unsigned *idp,typecode *code,unsigned *dcell
  ,double2 *posxy,double *posz,float4 *velrhop,float *auxnn,float4 *velrhopm1)
{
  if(n) {
    uint3 cellmax=make_uint3(domcells.x-1,domcells.y-1,domcells.z-1);
    dim3 sgrid=GetSimpleGridSize(n,SPHBSIZE);
    KerPeriodicDuplicateVerlet<<<sgrid,SPHBSIZE>>>(n,pini,cellmax,Double3(perinc),listp,idp,code,dcell,posxy,posz,velrhop,auxnn,velrhopm1);
  }
}
*/
//##############################################################################
//# Kernels for calculating NN Tensors.
//# Kernels para calcular tensores.
//##############################################################################
//------------------------------------------------------------------------------
/// Velocity gradients for non-Newtonian models using FDAs approach.
/// Gradientes de velocidad usando FDAs.
//------------------------------------------------------------------------------
/*
__device__ void GetVelocityGradients_FDA(float rr2,float drx,float dry,float drz
  ,float dvx,float dvy,float dvz,tmatrix3f &dvelp1,float &div_vel)
{
  //vel gradients
  dvelp1.a11=dvx*drx/rr2; dvelp1.a12=dvx*dry/rr2; dvelp1.a13=dvx*drz/rr2; //Fan et al., 2010
  dvelp1.a21=dvy*drx/rr2; dvelp1.a22=dvy*dry/rr2; dvelp1.a23=dvy*drz/rr2;
  dvelp1.a31=dvz*drx/rr2; dvelp1.a32=dvz*dry/rr2; dvelp1.a33=dvz*drz/rr2;
  div_vel=(dvelp1.a11+dvelp1.a22+dvelp1.a33)/3.f;
}
*/
//==============================================================================
//symetric tensors
//==============================================================================
/// Calculates the Stress Tensor (symetric)
/// Obtener tensor de velocidad de deformacion symetric.
//==============================================================================

__device__ void GetStressTensor_sym(float2 &d_p1_xx_xy,float2 &d_p1_xz_yy,float2 &d_p1_yz_zz,float visco_etap1
  ,float &I_t,float &II_t,float &J1_t,float &J2_t,float &tau_tensor_magn
  ,float2 &tau_xx_xy,float2 &tau_xz_yy,float2 &tau_yz_zz)
{
  //Stress tensor and invariant
  tau_xx_xy.x=2.f*visco_etap1*(d_p1_xx_xy.x);	tau_xx_xy.y=2.f*visco_etap1*d_p1_xx_xy.y;		tau_xz_yy.x=2.f*visco_etap1*d_p1_xz_yy.x;
  tau_xz_yy.y=2.f*visco_etap1*(d_p1_xz_yy.y);	tau_yz_zz.x=2.f*visco_etap1*d_p1_yz_zz.x;
  tau_yz_zz.y=2.f*visco_etap1*(d_p1_yz_zz.y);
  //I_t - the first invariant -
  I_t=tau_xx_xy.x+tau_xz_yy.y+tau_yz_zz.y;
  //II_t - the second invariant - expnaded form witout symetry 
  float II_t_1=tau_xx_xy.x*tau_xz_yy.y+tau_xz_yy.y*tau_yz_zz.y+tau_xx_xy.x*tau_yz_zz.y;
  float II_t_2=tau_xx_xy.y*tau_xx_xy.y+tau_yz_zz.x*tau_yz_zz.x+tau_xz_yy.x*tau_xz_yy.x;
  II_t=-II_t_1+II_t_2;
  //stress tensor magnitude
  tau_tensor_magn=sqrt(II_t);
  //if (II_t < 0.f) {
  //	printf("****tau_tensor_magn is negative**** \n");
  //}
  //Main Stress rate invariants
  J1_t=I_t; J2_t=I_t*I_t-2.f*II_t;
}

//==============================================================================
/// Calculates the Strain Rate Tensor (symetric).
/// Obtener tensor de velocidad de deformacion symetric.
//==============================================================================
__device__ void GetStrainRateTensor_tsym(float3 &dvelp1_xx_xy_xz,float3 &dvelp1_yx_yy_yz,float3 &dvelp1_zx_zy_zz
  ,float &I_D,float &II_D,float &J1_D,float &J2_D,float &div_D_tensor,float &D_tensor_magn
  ,float2 &D_tensor_xx_xy,float2 &D_tensor_xz_yy,float2 &D_tensor_yz_zz)
{
  //Strain tensor and invariant	
  float div_vel=(dvelp1_xx_xy_xz.x+dvelp1_yx_yy_yz.y+dvelp1_zx_zy_zz.z)/3.f;
  D_tensor_xx_xy.x=dvelp1_xx_xy_xz.x-div_vel;		D_tensor_xx_xy.y=0.5f*(dvelp1_xx_xy_xz.y+dvelp1_yx_yy_yz.x);		D_tensor_xz_yy.x=0.5f*(dvelp1_zx_zy_zz.x+dvelp1_xx_xy_xz.z);
  D_tensor_xz_yy.y=dvelp1_yx_yy_yz.y-div_vel;	  D_tensor_yz_zz.x=0.5f*(dvelp1_zx_zy_zz.y+dvelp1_yx_yy_yz.z);
  D_tensor_yz_zz.y=dvelp1_zx_zy_zz.z-div_vel;
  //the off-diagonal entries of velocity gradients are i.e. 0.5f*(du/dy+dvdx) with dvelp1.xy=du/dy+dvdx
  div_D_tensor=(D_tensor_xx_xy.x+D_tensor_xz_yy.y+D_tensor_yz_zz.y)/3.f;

  ////I_D - the first invariant -
  I_D=D_tensor_xx_xy.x+D_tensor_xz_yy.y+D_tensor_yz_zz.y;
  //II_D - the second invariant - expnaded form witout symetry 
  float II_D_1=D_tensor_xx_xy.x*D_tensor_xz_yy.y+D_tensor_xz_yy.y*D_tensor_yz_zz.y+D_tensor_xx_xy.x*D_tensor_yz_zz.y;
  float II_D_2=D_tensor_xx_xy.y*D_tensor_xx_xy.y+D_tensor_yz_zz.x*D_tensor_yz_zz.x+D_tensor_xz_yy.x*D_tensor_xz_yy.x;
  II_D=-II_D_1+II_D_2;
  ////deformation tensor magnitude
  D_tensor_magn=sqrt((II_D));

  //Main Strain rate invariants
  J1_D=I_D; J2_D=I_D*I_D-2.f*II_D;
}

//==============================================================================
/// Velocity gradients using SPH approach.
/// Gradientes de velocidad usando SPH.
//==============================================================================
__device__ void GetVelocityGradients_SPH_tsym(float massp2,const float4 &velrhop2,float dvx,float dvy,float dvz,float frx,float fry,float frz
  ,float3 &grap1_xx_xy_xz,float3 &grap1_yx_yy_yz,float3 &grap1_zx_zy_zz)
{
  ///SPH vel gradients calculation
  const float volp2=-massp2/velrhop2.w;
  float dv=dvx*volp2;  grap1_xx_xy_xz.x+=dv*frx;  grap1_xx_xy_xz.y+=dv*fry; grap1_xx_xy_xz.z+=dv*frz;
        dv=dvy*volp2;  grap1_yx_yy_yz.x+=dv*frx;	grap1_yx_yy_yz.y+=dv*fry; grap1_yx_yy_yz.z+=dv*frz;
        dv=dvz*volp2;  grap1_zx_zy_zz.x+=dv*frx;  grap1_zx_zy_zz.y+=dv*fry; grap1_zx_zy_zz.z+=dv*frz;
}

//==============================================================================
/// Calculate strain rate tensor (full matrix).
/// Obtener tensor de velocidad de deformacion (full matrix).
//==============================================================================
/*
__device__ void GetStrainRateTensor(const tmatrix3f &dvelp1,float div_vel,float &I_D,float &II_D,float &J1_D
  ,float &J2_D,float &div_D_tensor,float &D_tensor_magn,tmatrix3f &D_tensor)
{
  //Strain tensor and invariant
  D_tensor.a11=dvelp1.a11-div_vel;          D_tensor.a12=0.5f*(dvelp1.a12+dvelp1.a21);      D_tensor.a13=0.5f*(dvelp1.a13+dvelp1.a31);
  D_tensor.a21=0.5f*(dvelp1.a21+dvelp1.a12);      D_tensor.a22=dvelp1.a22-div_vel;          D_tensor.a23=0.5f*(dvelp1.a23+dvelp1.a32);
  D_tensor.a31=0.5f*(dvelp1.a31+dvelp1.a13);      D_tensor.a32=0.5f*(dvelp1.a32+dvelp1.a23);      D_tensor.a33=dvelp1.a33-div_vel;
  div_D_tensor=(D_tensor.a11+D_tensor.a22+D_tensor.a33)/3.f;

  //I_D - the first invariant -
  I_D=D_tensor.a11+D_tensor.a22+D_tensor.a33;
  //II_D - the second invariant - expnaded form witout symetry 
  float II_D_1=D_tensor.a11*D_tensor.a22+D_tensor.a22*D_tensor.a33+D_tensor.a11*D_tensor.a33;
  float II_D_2=D_tensor.a12*D_tensor.a21+D_tensor.a23*D_tensor.a32+D_tensor.a13*D_tensor.a31;
  II_D=II_D_1-II_D_2;
  //deformation tensor magnitude
  D_tensor_magn=sqrt((II_D*II_D));

  //Main Strain rate invariants
  J1_D=I_D; J2_D=I_D*I_D-2.f*II_D;
}
*/
//==============================================================================
/// Calculate strain rate tensor and the spin rate tensor (full matrix).
//==============================================================================
__device__ void GetStrainSpinRateTensor(float3 &dvelp1_xx_xy_xz,float3 &dvelp1_yx_yy_yz,float3 &dvelp1_zx_zy_zz
  ,float2 &D_tensor_xx_xy,float2 &D_tensor_xz_yy,float2 &D_tensor_yz_zz, float3 &W_tensor_xyz)
{
  //Strain tensor
  D_tensor_xx_xy.x=dvelp1_xx_xy_xz.x;		
  D_tensor_xz_yy.y=dvelp1_yx_yy_yz.y;	  
  D_tensor_yz_zz.y=dvelp1_zx_zy_zz.z;
  D_tensor_xx_xy.y=0.5f*(dvelp1_xx_xy_xz.y+dvelp1_yx_yy_yz.x);
  D_tensor_xz_yy.x=0.5f*(dvelp1_xx_xy_xz.z+dvelp1_zx_zy_zz.x);
  D_tensor_yz_zz.x=0.5f*(dvelp1_yx_yy_yz.z+dvelp1_zx_zy_zz.y);

  //Full spin tensor
  W_tensor_xyz.x = 0.5f*(dvelp1_xx_xy_xz.y-dvelp1_yx_yy_yz.x);
  W_tensor_xyz.y = 0.5f*(dvelp1_xx_xy_xz.z-dvelp1_zx_zy_zz.x);
  W_tensor_xyz.z = 0.5f*(dvelp1_yx_yy_yz.z-dvelp1_zx_zy_zz.y);
}

//==============================================================================
/// Calculates the stress Tensor from hypoplastic model
//==============================================================================
__device__ void GetStressTensorHypo(float2 &dtsrp1_xx_xy, float2 &dtsrp1_xz_yy, float2 &dtsrp1_yz_zz, float3 &dtspinrate_xyz
  ,float2 &taup1_xx_xy_old, float2 &taup1_xz_yy_old, float2 &taup1_yz_zz_old
  ,float2 &taup1_xx_xy, float2 &taup1_xz_yy, float2 &taup1_yz_zz
  ,float2 &taup1_diff_xx_xy, float2 &taup1_diff_xz_yy, float2 &taup1_diff_yz_zz
  ,float &voidRatio, double &dt, const float &Hypo_angle, const float &Hypo_hs, const float &Hypo_n
  ,const float &Hypo_ed0, const float &Hypo_ec0, const float &Hypo_ei0, const float &Hypo_alpha, const float &Hypo_beta, bool regularize, bool stop)
{
  tmatrix3f D_tensor = {dtsrp1_xx_xy.x, dtsrp1_xx_xy.y, dtsrp1_xz_yy.x, 
                        dtsrp1_xx_xy.y, dtsrp1_xz_yy.y, dtsrp1_yz_zz.x, 
                        dtsrp1_xz_yy.x, dtsrp1_yz_zz.x, dtsrp1_yz_zz.y};
  tmatrix3f W_tensor = {0, dtspinrate_xyz.x, dtspinrate_xyz.y,
                        -dtspinrate_xyz.x, 0, dtspinrate_xyz.z,
                        -dtspinrate_xyz.y, -dtspinrate_xyz.z,0};
  tmatrix3f SigmaOld_tensor= {taup1_xx_xy_old.x,  taup1_xx_xy_old.y, taup1_xz_yy_old.x, 
                              taup1_xx_xy_old.y,  taup1_xz_yy_old.y, taup1_yz_zz_old.x, 
                              taup1_xz_yy_old.x,  taup1_yz_zz_old.x, taup1_yz_zz_old.y};

  float TrSigma = taup1_xx_xy_old.x+taup1_xz_yy_old.y+taup1_yz_zz_old.y; //Sigma_kk
  // convert stress vector to full stress tensor
  tmatrix3f Ts_tensor;
  Ts_tensor.a11 = SigmaOld_tensor.a11/TrSigma;
  Ts_tensor.a22 = SigmaOld_tensor.a22/TrSigma;
  Ts_tensor.a33 = SigmaOld_tensor.a33/TrSigma;
  Ts_tensor.a12 = SigmaOld_tensor.a12/TrSigma;
  Ts_tensor.a21 = SigmaOld_tensor.a21/TrSigma;
  Ts_tensor.a13 = SigmaOld_tensor.a13/TrSigma;
  Ts_tensor.a31 = SigmaOld_tensor.a31/TrSigma;
  Ts_tensor.a23 = SigmaOld_tensor.a23/TrSigma;
  Ts_tensor.a32 = SigmaOld_tensor.a32/TrSigma;
  
  tmatrix3f Ts2_tensor;
  Ts2_tensor.a11 = Ts_tensor.a11*Ts_tensor.a11+Ts_tensor.a12*Ts_tensor.a21+Ts_tensor.a13*Ts_tensor.a31;
  Ts2_tensor.a12 = Ts_tensor.a11*Ts_tensor.a12+Ts_tensor.a12*Ts_tensor.a22+Ts_tensor.a13*Ts_tensor.a32;
  Ts2_tensor.a13 = Ts_tensor.a11*Ts_tensor.a13+Ts_tensor.a12*Ts_tensor.a23+Ts_tensor.a13*Ts_tensor.a33;
  Ts2_tensor.a21 = Ts_tensor.a21*Ts_tensor.a11+Ts_tensor.a22*Ts_tensor.a21+Ts_tensor.a23*Ts_tensor.a31;
  Ts2_tensor.a22 = Ts_tensor.a21*Ts_tensor.a12+Ts_tensor.a22*Ts_tensor.a22+Ts_tensor.a23*Ts_tensor.a32;
  Ts2_tensor.a23 = Ts_tensor.a21*Ts_tensor.a13+Ts_tensor.a22*Ts_tensor.a23+Ts_tensor.a23*Ts_tensor.a33;
  Ts2_tensor.a31 = Ts_tensor.a31*Ts_tensor.a11+Ts_tensor.a32*Ts_tensor.a21+Ts_tensor.a33*Ts_tensor.a31;
  Ts2_tensor.a32 = Ts_tensor.a31*Ts_tensor.a12+Ts_tensor.a32*Ts_tensor.a22+Ts_tensor.a33*Ts_tensor.a32;
  Ts2_tensor.a33 = Ts_tensor.a31*Ts_tensor.a13+Ts_tensor.a32*Ts_tensor.a23+Ts_tensor.a33*Ts_tensor.a33;
  float TrTs2 =Ts2_tensor.a11+Ts2_tensor.a22+Ts2_tensor.a33;

  tmatrix3f Tsv_tensor; //normalized deviatoric stress tensor sigmaHeadStar_ij
  float OneOverThree = 1.0/3.0;
  Tsv_tensor.a11 = Ts_tensor.a11 - OneOverThree;
  Tsv_tensor.a12 = Ts_tensor.a12;
  Tsv_tensor.a13 = Ts_tensor.a13;
  Tsv_tensor.a21 = Ts_tensor.a21;
  Tsv_tensor.a22 = Ts_tensor.a22 - OneOverThree;
  Tsv_tensor.a23 = Ts_tensor.a23;
  Tsv_tensor.a31 = Ts_tensor.a31;
  Tsv_tensor.a32 = Ts_tensor.a32;
  Tsv_tensor.a33 = Ts_tensor.a33 - OneOverThree;

  tmatrix3f Tsv2_tensor, Tsv3_tensor;
  Tsv2_tensor.a11 = Tsv_tensor.a11*Tsv_tensor.a11+Tsv_tensor.a12*Tsv_tensor.a21+Tsv_tensor.a13*Tsv_tensor.a31;
  Tsv2_tensor.a12 = Tsv_tensor.a11*Tsv_tensor.a12+Tsv_tensor.a12*Tsv_tensor.a22+Tsv_tensor.a13*Tsv_tensor.a32;
  Tsv2_tensor.a13 = Tsv_tensor.a11*Tsv_tensor.a13+Tsv_tensor.a12*Tsv_tensor.a23+Tsv_tensor.a13*Tsv_tensor.a33;
  Tsv2_tensor.a21 = Tsv_tensor.a21*Tsv_tensor.a11+Tsv_tensor.a22*Tsv_tensor.a21+Tsv_tensor.a23*Tsv_tensor.a31;
  Tsv2_tensor.a22 = Tsv_tensor.a21*Tsv_tensor.a12+Tsv_tensor.a22*Tsv_tensor.a22+Tsv_tensor.a23*Tsv_tensor.a32;
  Tsv2_tensor.a23 = Tsv_tensor.a21*Tsv_tensor.a13+Tsv_tensor.a22*Tsv_tensor.a23+Tsv_tensor.a23*Tsv_tensor.a33;
  Tsv2_tensor.a31 = Tsv_tensor.a31*Tsv_tensor.a11+Tsv_tensor.a32*Tsv_tensor.a21+Tsv_tensor.a33*Tsv_tensor.a31;
  Tsv2_tensor.a32 = Tsv_tensor.a31*Tsv_tensor.a12+Tsv_tensor.a32*Tsv_tensor.a22+Tsv_tensor.a33*Tsv_tensor.a32;
  Tsv2_tensor.a33 = Tsv_tensor.a31*Tsv_tensor.a13+Tsv_tensor.a32*Tsv_tensor.a23+Tsv_tensor.a33*Tsv_tensor.a33;

  Tsv3_tensor.a11 = Tsv2_tensor.a11*Tsv_tensor.a11+Tsv2_tensor.a12*Tsv_tensor.a21+Tsv2_tensor.a13*Tsv_tensor.a31;
  Tsv3_tensor.a12 = Tsv2_tensor.a11*Tsv_tensor.a12+Tsv2_tensor.a12*Tsv_tensor.a22+Tsv2_tensor.a13*Tsv_tensor.a32;
  Tsv3_tensor.a13 = Tsv2_tensor.a11*Tsv_tensor.a13+Tsv2_tensor.a12*Tsv_tensor.a23+Tsv2_tensor.a13*Tsv_tensor.a33;
  Tsv3_tensor.a21 = Tsv2_tensor.a21*Tsv_tensor.a11+Tsv2_tensor.a22*Tsv_tensor.a21+Tsv2_tensor.a23*Tsv_tensor.a31;
  Tsv3_tensor.a22 = Tsv2_tensor.a21*Tsv_tensor.a12+Tsv2_tensor.a22*Tsv_tensor.a22+Tsv2_tensor.a23*Tsv_tensor.a32;
  Tsv3_tensor.a23 = Tsv2_tensor.a21*Tsv_tensor.a13+Tsv2_tensor.a22*Tsv_tensor.a23+Tsv2_tensor.a23*Tsv_tensor.a33;
  Tsv3_tensor.a31 = Tsv2_tensor.a31*Tsv_tensor.a11+Tsv2_tensor.a32*Tsv_tensor.a21+Tsv2_tensor.a33*Tsv_tensor.a31;
  Tsv3_tensor.a32 = Tsv2_tensor.a31*Tsv_tensor.a12+Tsv2_tensor.a32*Tsv_tensor.a22+Tsv2_tensor.a33*Tsv_tensor.a32;
  Tsv3_tensor.a33 = Tsv2_tensor.a31*Tsv_tensor.a13+Tsv2_tensor.a32*Tsv_tensor.a23+Tsv2_tensor.a33*Tsv_tensor.a33;

  float TrTsv2 = Tsv2_tensor.a11+Tsv2_tensor.a22+Tsv2_tensor.a33;
  float TrTsv3 = Tsv3_tensor.a11+Tsv3_tensor.a22+Tsv3_tensor.a33;
  float sinphi = sin(Hypo_angle*PI/180);
  float sq2 = sqrt(2.0);
  float sq3 = sqrt(3.0);
  float sq6 = sqrt(6.0);
  float c3t; //Lode angle cos(3*theta)
  if (TrTsv2<=1E-10){
    c3t = 1.0;
  }else{
    c3t = -sq6*TrTsv3/pow(TrTsv2, 1.5);
    if (c3t > 1.0)  c3t =  1;
    if (c3t < -1.0) c3t = -1;
  }
  float c1 = sq3*( 3-sinphi )/( 2*sq2*sinphi );
  float c2 = 3*( 3+sinphi )/( 8*sinphi );
  float a1 = 1/(c1+c2*sqrt(TrTsv2)*(1+c3t));

  float ed = Hypo_ed0*exp(-pow((-TrSigma/Hypo_hs),Hypo_n));
  float ec = Hypo_ec0*exp(-pow((-TrSigma/Hypo_hs),Hypo_n));
  float ei = Hypo_ei0*exp(-pow((-TrSigma/Hypo_hs),Hypo_n));
  
  //if (voidRatio<ed){
    //Log->Printf("void ratio e = %f is less than the minimum void ratio at current stress ed = %f ", voidRatio, ed);
    //printf(">> current e=%f  current ed=%f\n",voidRatio,ed);
    //voidRatio = ed;
  //}
   
  if (voidRatio>ei){
    //Log->Printf("void ratio e = %f is more than the maximum void ratio at current stress ei = %f ", voidRatio, ei);
    voidRatio = ei;
  } 

  float fe = pow(ec/voidRatio,Hypo_beta);
  float hi = 1/pow(c1, 2) + OneOverThree - pow((Hypo_ei0-Hypo_ed0)/(Hypo_ec0-Hypo_ed0),Hypo_alpha)/c1/sq3; 
  //float fb = Hypo_hs / Hypo_n / hi * (1 + ei) / ei * pow(-TrSigma / Hypo_hs, 1 - Hypo_n); 
  float fb = Hypo_hs / Hypo_n / hi * (1 + ei) / ei *pow(Hypo_ei0/Hypo_ec0,Hypo_beta)* pow(-TrSigma / Hypo_hs, 1 - Hypo_n); 
  float term5 = Ts_tensor.a11*D_tensor.a11 + Ts_tensor.a12*D_tensor.a12 + Ts_tensor.a13*D_tensor.a13 + Ts_tensor.a21*D_tensor.a21 + Ts_tensor.a22*D_tensor.a22 + Ts_tensor.a23*D_tensor.a23 + Ts_tensor.a31*D_tensor.a31 + Ts_tensor.a32*D_tensor.a32 + Ts_tensor.a33*D_tensor.a33;
  float term6 = sqrt(D_tensor.a11*D_tensor.a11 + D_tensor.a12*D_tensor.a12 + D_tensor.a13*D_tensor.a13 + D_tensor.a21*D_tensor.a21 + D_tensor.a22*D_tensor.a22 + D_tensor.a23*D_tensor.a23 + D_tensor.a31*D_tensor.a31 + D_tensor.a32*D_tensor.a32 + D_tensor.a33*D_tensor.a33);

  float fs = fb*fe;
  float fd = ( voidRatio-ed )/( ec-ed );
  if (fd>0){
    fd=pow(fd, Hypo_alpha);
  }else{
    fd=0;
  }

  tmatrix3f FLL, FNN, sigma_rate, tau_tensor;
  FLL.a11 = pow(a1,2)*D_tensor.a11 + term5*Ts_tensor.a11;
  FLL.a12 = pow(a1,2)*D_tensor.a12 + term5*Ts_tensor.a12;
  FLL.a13 = pow(a1,2)*D_tensor.a13 + term5*Ts_tensor.a13;
  FLL.a21 = pow(a1,2)*D_tensor.a21 + term5*Ts_tensor.a21;
  FLL.a22 = pow(a1,2)*D_tensor.a22 + term5*Ts_tensor.a22;
  FLL.a23 = pow(a1,2)*D_tensor.a23 + term5*Ts_tensor.a23;
  FLL.a31 = pow(a1,2)*D_tensor.a31 + term5*Ts_tensor.a31;
  FLL.a32 = pow(a1,2)*D_tensor.a32 + term5*Ts_tensor.a32;
  FLL.a33 = pow(a1,2)*D_tensor.a33 + term5*Ts_tensor.a33;

  FNN.a11  = a1*(Ts_tensor.a11+Tsv_tensor.a11);
  FNN.a12  = a1*(Ts_tensor.a12+Tsv_tensor.a12);
  FNN.a13  = a1*(Ts_tensor.a13+Tsv_tensor.a13);
  FNN.a21  = a1*(Ts_tensor.a21+Tsv_tensor.a21);
  FNN.a22  = a1*(Ts_tensor.a22+Tsv_tensor.a22);
  FNN.a23  = a1*(Ts_tensor.a23+Tsv_tensor.a23);
  FNN.a31  = a1*(Ts_tensor.a31+Tsv_tensor.a31);
  FNN.a32  = a1*(Ts_tensor.a32+Tsv_tensor.a32);
  FNN.a33  = a1*(Ts_tensor.a33+Tsv_tensor.a33);

  //Stress tensor and invariant   
  sigma_rate.a11 = fs*(FLL.a11+fd*term6*FNN.a11);
  sigma_rate.a12 = fs*(FLL.a12+fd*term6*FNN.a12);
  sigma_rate.a13 = fs*(FLL.a13+fd*term6*FNN.a13);
  sigma_rate.a21 = fs*(FLL.a21+fd*term6*FNN.a21);
  sigma_rate.a22 = fs*(FLL.a22+fd*term6*FNN.a22);
  sigma_rate.a23 = fs*(FLL.a23+fd*term6*FNN.a23);
  sigma_rate.a31 = fs*(FLL.a31+fd*term6*FNN.a31);
  sigma_rate.a32 = fs*(FLL.a32+fd*term6*FNN.a32);
  sigma_rate.a33 = fs*(FLL.a33+fd*term6*FNN.a33);

  //Rotation
  sigma_rate.a11 = sigma_rate.a11 + (W_tensor.a11*SigmaOld_tensor.a11+W_tensor.a12*SigmaOld_tensor.a21+W_tensor.a13*SigmaOld_tensor.a31) - (SigmaOld_tensor.a11*W_tensor.a11+SigmaOld_tensor.a12*W_tensor.a21+SigmaOld_tensor.a13*W_tensor.a31);
  sigma_rate.a12 = sigma_rate.a12 + (W_tensor.a11*SigmaOld_tensor.a12+W_tensor.a12*SigmaOld_tensor.a22+W_tensor.a13*SigmaOld_tensor.a32) - (SigmaOld_tensor.a11*W_tensor.a12+SigmaOld_tensor.a12*W_tensor.a22+SigmaOld_tensor.a13*W_tensor.a32);
  sigma_rate.a13 = sigma_rate.a13 + (W_tensor.a11*SigmaOld_tensor.a13+W_tensor.a12*SigmaOld_tensor.a23+W_tensor.a13*SigmaOld_tensor.a33) - (SigmaOld_tensor.a11*W_tensor.a13+SigmaOld_tensor.a12*W_tensor.a23+SigmaOld_tensor.a13*W_tensor.a33);
  sigma_rate.a21 = sigma_rate.a21 + (W_tensor.a21*SigmaOld_tensor.a11+W_tensor.a22*SigmaOld_tensor.a21+W_tensor.a23*SigmaOld_tensor.a31) - (SigmaOld_tensor.a21*W_tensor.a11+SigmaOld_tensor.a22*W_tensor.a21+SigmaOld_tensor.a23*W_tensor.a31);
  sigma_rate.a22 = sigma_rate.a22 + (W_tensor.a21*SigmaOld_tensor.a12+W_tensor.a22*SigmaOld_tensor.a22+W_tensor.a23*SigmaOld_tensor.a32) - (SigmaOld_tensor.a21*W_tensor.a12+SigmaOld_tensor.a22*W_tensor.a22+SigmaOld_tensor.a23*W_tensor.a32);
  sigma_rate.a23 = sigma_rate.a23 + (W_tensor.a21*SigmaOld_tensor.a13+W_tensor.a22*SigmaOld_tensor.a23+W_tensor.a23*SigmaOld_tensor.a33) - (SigmaOld_tensor.a21*W_tensor.a13+SigmaOld_tensor.a22*W_tensor.a23+SigmaOld_tensor.a23*W_tensor.a33);
  sigma_rate.a31 = sigma_rate.a31 + (W_tensor.a31*SigmaOld_tensor.a11+W_tensor.a32*SigmaOld_tensor.a21+W_tensor.a33*SigmaOld_tensor.a31) - (SigmaOld_tensor.a31*W_tensor.a11+SigmaOld_tensor.a32*W_tensor.a21+SigmaOld_tensor.a33*W_tensor.a31);
  sigma_rate.a32 = sigma_rate.a32 + (W_tensor.a31*SigmaOld_tensor.a12+W_tensor.a32*SigmaOld_tensor.a22+W_tensor.a33*SigmaOld_tensor.a32) - (SigmaOld_tensor.a31*W_tensor.a12+SigmaOld_tensor.a32*W_tensor.a22+SigmaOld_tensor.a33*W_tensor.a32);
  sigma_rate.a33 = sigma_rate.a33 + (W_tensor.a31*SigmaOld_tensor.a13+W_tensor.a32*SigmaOld_tensor.a23+W_tensor.a33*SigmaOld_tensor.a33) - (SigmaOld_tensor.a31*W_tensor.a13+SigmaOld_tensor.a32*W_tensor.a23+SigmaOld_tensor.a33*W_tensor.a33);

  tau_tensor.a11 = SigmaOld_tensor.a11 + sigma_rate.a11*dt;
  tau_tensor.a12 = SigmaOld_tensor.a12 + sigma_rate.a12*dt;
  tau_tensor.a13 = SigmaOld_tensor.a13 + sigma_rate.a13*dt;
  tau_tensor.a21 = SigmaOld_tensor.a21 + sigma_rate.a21*dt;
  tau_tensor.a22 = SigmaOld_tensor.a22 + sigma_rate.a22*dt;
  tau_tensor.a23 = SigmaOld_tensor.a23 + sigma_rate.a23*dt;
  tau_tensor.a31 = SigmaOld_tensor.a31 + sigma_rate.a31*dt;
  tau_tensor.a32 = SigmaOld_tensor.a32 + sigma_rate.a32*dt;
  tau_tensor.a33 = SigmaOld_tensor.a33 + sigma_rate.a33*dt;
  /*
  if(isnan(tau_tensor.a11) || isinf(tau_tensor.a11)  || abs(sigma_rate.a11*dt/SigmaOld_tensor.a11)>0.5){
    tau_tensor.a11 = SigmaOld_tensor.a11;
  }
  if(isnan(tau_tensor.a12) || isinf(tau_tensor.a12) || abs(sigma_rate.a12*dt/SigmaOld_tensor.a12)>0.5){
    tau_tensor.a12 = SigmaOld_tensor.a12;
  }
  if(isnan(tau_tensor.a13) || isinf(tau_tensor.a13) || abs(sigma_rate.a13*dt/SigmaOld_tensor.a13)>0.5){
    tau_tensor.a13 = SigmaOld_tensor.a13;
  }
  if(isnan(tau_tensor.a21) || isinf(tau_tensor.a21) || abs(sigma_rate.a21*dt/SigmaOld_tensor.a21)>0.5){
    tau_tensor.a21 = SigmaOld_tensor.a21;
  }
  if(isnan(tau_tensor.a22) || isinf(tau_tensor.a22) || abs(sigma_rate.a22*dt/SigmaOld_tensor.a22)>0.5){
    tau_tensor.a22 = SigmaOld_tensor.a22;
  }
  if(isnan(tau_tensor.a23) || isinf(tau_tensor.a23) || abs(sigma_rate.a23*dt/SigmaOld_tensor.a23)>0.5){
    tau_tensor.a23 = SigmaOld_tensor.a23;
  }
  if(isnan(tau_tensor.a31) || isinf(tau_tensor.a31) || abs(sigma_rate.a31*dt/SigmaOld_tensor.a31)>0.5){
    tau_tensor.a31 = SigmaOld_tensor.a31;
  }
  if(isnan(tau_tensor.a32) || isinf(tau_tensor.a32) || abs(sigma_rate.a32*dt/SigmaOld_tensor.a32)>0.5){
    tau_tensor.a32 = SigmaOld_tensor.a32;
  }
  if(isnan(tau_tensor.a33) || isinf(tau_tensor.a33) || abs(sigma_rate.a33*dt/SigmaOld_tensor.a33)>0.5){
    tau_tensor.a33 = SigmaOld_tensor.a33;
  }
  */
  taup1_xx_xy.x = tau_tensor.a11;
  taup1_xx_xy.y = 0.5f * (tau_tensor.a12 + tau_tensor.a21);
  taup1_xz_yy.x = 0.5f * (tau_tensor.a13 + tau_tensor.a31);
  taup1_yz_zz.x = 0.5f * (tau_tensor.a23 + tau_tensor.a32);
  taup1_xz_yy.y = tau_tensor.a22;
  taup1_yz_zz.y = tau_tensor.a33;

  if (tau_tensor.a11 > 0){
    taup1_xx_xy.x = -10;
    taup1_xx_xy.y = 0;
    taup1_xz_yy.x = 0;
  } else{
    taup1_xx_xy.x = tau_tensor.a11;
    taup1_xx_xy.y = 0.5f*(tau_tensor.a12+tau_tensor.a21);
    taup1_xz_yy.x = 0.5f*(tau_tensor.a13+tau_tensor.a31);
  }
  if (tau_tensor.a22 > 0){
    taup1_yz_zz.x = 0;
    taup1_xz_yy.y = -10;
  }else{
    taup1_yz_zz.x = 0.5f*(tau_tensor.a23+tau_tensor.a32);
    taup1_xz_yy.y = tau_tensor.a22;
  }
  if (tau_tensor.a33 > 0){
    taup1_yz_zz.y = -10;
  }else{
    taup1_yz_zz.y = tau_tensor.a33;
  }

  voidRatio = voidRatio + (1+voidRatio)*(D_tensor.a11 + D_tensor.a22 + D_tensor.a33)*dt  ;

//  check whether void ratio is outside allowed range 
  TrSigma=taup1_xx_xy.x+taup1_xz_yy.y+taup1_yz_zz.y;
//  float edd = Hypo_ed0*exp(-pow((-TrSigma/Hypo_hs),Hypo_n))*1.001;
  float eii = Hypo_ei0*exp(-pow((-TrSigma/Hypo_hs),Hypo_n))*0.999;
//  if (voidRatio<edd) voidRatio=edd;
  if (voidRatio>eii) voidRatio=eii;

  if(regularize){
    taup1_xx_xy.x +=taup1_diff_xx_xy.x*dt;
    taup1_xx_xy.y +=taup1_diff_xx_xy.y*dt;
    taup1_xz_yy.x +=taup1_diff_xz_yy.x*dt;
    taup1_xz_yy.y +=taup1_diff_xz_yy.y*dt;
    taup1_yz_zz.x +=taup1_diff_yz_zz.x*dt;
    taup1_yz_zz.y +=taup1_diff_yz_zz.y*dt;
  }
}

//==============================================================================
/// Calculates the stress Tensor from elastic model
//==============================================================================
__device__ void GetStressTensorElastic(float2 &dtsrp1_xx_xy, float2 &dtsrp1_xz_yy, float2 &dtsrp1_yz_zz, float3 &dtspinrate_xyz
  ,float2 &taup1_xx_xy_old, float2 &taup1_xz_yy_old, float2 &taup1_yz_zz_old
  ,float2 &taup1_diff_xx_xy, float2 &taup1_diff_xz_yy, float2 &taup1_diff_yz_zz
  ,float2 &taup1_xx_xy, float2 &taup1_xz_yy, float2 &taup1_yz_zz
  ,double &dt, const float &lameparm2, const float &lameparm1, bool regularize)
{
  tmatrix3f W_tensor = {0, dtspinrate_xyz.x, dtspinrate_xyz.y,
                      -dtspinrate_xyz.x, 0, dtspinrate_xyz.z,
                      -dtspinrate_xyz.y, -dtspinrate_xyz.z,0};
  tmatrix3f SigmaOld_tensor= {taup1_xx_xy_old.x,  taup1_xx_xy_old.y, taup1_xz_yy_old.x, 
                              taup1_xx_xy_old.y,  taup1_xz_yy_old.y, taup1_yz_zz_old.x, 
                              taup1_xz_yy_old.x,  taup1_yz_zz_old.x, taup1_yz_zz_old.y};
  // sigma_rate = lameparm1 * Identity ^ TrD_tensor + 2 * lameparm2 * D_tensor
  float TrD_tensor = dtsrp1_xx_xy.x + dtsrp1_xz_yy.y + dtsrp1_yz_zz.y; //D_tensor_kk
  tmatrix3f sigma_rate, tau_tensor;
  sigma_rate.a11 = lameparm1 * TrD_tensor + 2 * lameparm2 * dtsrp1_xx_xy.x;
  sigma_rate.a12 = 2 * lameparm2 * dtsrp1_xx_xy.y;
  sigma_rate.a13 = 2 * lameparm2 * dtsrp1_xz_yy.x;
  sigma_rate.a21 = 2 * lameparm2 * dtsrp1_xx_xy.y;
  sigma_rate.a22 = lameparm1 * TrD_tensor + 2 * lameparm2 * dtsrp1_xz_yy.y;
  sigma_rate.a23 = 2 * lameparm2 * dtsrp1_yz_zz.x;
  sigma_rate.a31 = 2 * lameparm2 * dtsrp1_xz_yy.x;
  sigma_rate.a32 = 2 * lameparm2 * dtsrp1_yz_zz.x;
  sigma_rate.a33 = lameparm1 * TrD_tensor + 2 * lameparm2 * dtsrp1_yz_zz.y;
  
   //Rotation
  sigma_rate.a11 = sigma_rate.a11 + (W_tensor.a11*SigmaOld_tensor.a11+W_tensor.a12*SigmaOld_tensor.a21+W_tensor.a13*SigmaOld_tensor.a31) - (SigmaOld_tensor.a11*W_tensor.a11+SigmaOld_tensor.a12*W_tensor.a21+SigmaOld_tensor.a13*W_tensor.a31);
  sigma_rate.a12 = sigma_rate.a12 + (W_tensor.a11*SigmaOld_tensor.a12+W_tensor.a12*SigmaOld_tensor.a22+W_tensor.a13*SigmaOld_tensor.a32) - (SigmaOld_tensor.a11*W_tensor.a12+SigmaOld_tensor.a12*W_tensor.a22+SigmaOld_tensor.a13*W_tensor.a32);
  sigma_rate.a13 = sigma_rate.a13 + (W_tensor.a11*SigmaOld_tensor.a13+W_tensor.a12*SigmaOld_tensor.a23+W_tensor.a13*SigmaOld_tensor.a33) - (SigmaOld_tensor.a11*W_tensor.a13+SigmaOld_tensor.a12*W_tensor.a23+SigmaOld_tensor.a13*W_tensor.a33);
  sigma_rate.a21 = sigma_rate.a21 + (W_tensor.a21*SigmaOld_tensor.a11+W_tensor.a22*SigmaOld_tensor.a21+W_tensor.a23*SigmaOld_tensor.a31) - (SigmaOld_tensor.a21*W_tensor.a11+SigmaOld_tensor.a22*W_tensor.a21+SigmaOld_tensor.a23*W_tensor.a31);
  sigma_rate.a22 = sigma_rate.a22 + (W_tensor.a21*SigmaOld_tensor.a12+W_tensor.a22*SigmaOld_tensor.a22+W_tensor.a23*SigmaOld_tensor.a32) - (SigmaOld_tensor.a21*W_tensor.a12+SigmaOld_tensor.a22*W_tensor.a22+SigmaOld_tensor.a23*W_tensor.a32);
  sigma_rate.a23 = sigma_rate.a23 + (W_tensor.a21*SigmaOld_tensor.a13+W_tensor.a22*SigmaOld_tensor.a23+W_tensor.a23*SigmaOld_tensor.a33) - (SigmaOld_tensor.a21*W_tensor.a13+SigmaOld_tensor.a22*W_tensor.a23+SigmaOld_tensor.a23*W_tensor.a33);
  sigma_rate.a31 = sigma_rate.a31 + (W_tensor.a31*SigmaOld_tensor.a11+W_tensor.a32*SigmaOld_tensor.a21+W_tensor.a33*SigmaOld_tensor.a31) - (SigmaOld_tensor.a31*W_tensor.a11+SigmaOld_tensor.a32*W_tensor.a21+SigmaOld_tensor.a33*W_tensor.a31);
  sigma_rate.a32 = sigma_rate.a32 + (W_tensor.a31*SigmaOld_tensor.a12+W_tensor.a32*SigmaOld_tensor.a22+W_tensor.a33*SigmaOld_tensor.a32) - (SigmaOld_tensor.a31*W_tensor.a12+SigmaOld_tensor.a32*W_tensor.a22+SigmaOld_tensor.a33*W_tensor.a32);
  sigma_rate.a33 = sigma_rate.a33 + (W_tensor.a31*SigmaOld_tensor.a13+W_tensor.a32*SigmaOld_tensor.a23+W_tensor.a33*SigmaOld_tensor.a33) - (SigmaOld_tensor.a31*W_tensor.a13+SigmaOld_tensor.a32*W_tensor.a23+SigmaOld_tensor.a33*W_tensor.a33);

  tau_tensor.a11 = SigmaOld_tensor.a11 + sigma_rate.a11*dt;
  tau_tensor.a12 = SigmaOld_tensor.a12 + sigma_rate.a12*dt;
  tau_tensor.a13 = SigmaOld_tensor.a13 + sigma_rate.a13*dt;
  tau_tensor.a21 = SigmaOld_tensor.a21 + sigma_rate.a21*dt;
  tau_tensor.a22 = SigmaOld_tensor.a22 + sigma_rate.a22*dt;
  tau_tensor.a23 = SigmaOld_tensor.a23 + sigma_rate.a23*dt;
  tau_tensor.a31 = SigmaOld_tensor.a31 + sigma_rate.a31*dt;
  tau_tensor.a32 = SigmaOld_tensor.a32 + sigma_rate.a32*dt;
  tau_tensor.a33 = SigmaOld_tensor.a33 + sigma_rate.a33*dt;

  taup1_xx_xy.x = tau_tensor.a11;
  taup1_xx_xy.y = 0.5f * (tau_tensor.a12 + tau_tensor.a21);
  taup1_xz_yy.x = 0.5f * (tau_tensor.a13 + tau_tensor.a31);
  taup1_yz_zz.x = 0.5f * (tau_tensor.a23 + tau_tensor.a32);
  taup1_xz_yy.y = tau_tensor.a22;
  taup1_yz_zz.y = tau_tensor.a33;

  if(regularize){
    taup1_xx_xy.x +=taup1_diff_xx_xy.x*dt;
    taup1_xx_xy.y +=taup1_diff_xx_xy.y*dt;
    taup1_xz_yy.x +=taup1_diff_xz_yy.x*dt;
    taup1_xz_yy.y +=taup1_diff_xz_yy.y*dt;
    taup1_yz_zz.x +=taup1_diff_yz_zz.x*dt;
    taup1_yz_zz.y +=taup1_diff_yz_zz.y*dt;
  }

}


//==============================================================================
/// Calculates the effective visocity.
/// Calcule la viscosidad efectiva.
//==============================================================================
__device__ void KerGetEta_Effective(const typecode ppx,float tau_yield,float D_tensor_magn,float visco
  ,float m_NN,float n_NN,float &visco_etap1)
{

  if(D_tensor_magn<=ALMOSTZERO)D_tensor_magn=ALMOSTZERO;
  float miou_yield=(PHASECTE[ppx].tau_max ? PHASECTE[ppx].tau_max/(2.0f*D_tensor_magn) : (tau_yield)/(2.0f*D_tensor_magn)); //HPB will adjust eta		

  //if tau_max exists
  bool bi_region=PHASECTE[ppx].tau_max && D_tensor_magn<=PHASECTE[ppx].tau_max/(2.f*PHASECTE[ppx].Bi_multi*visco);
  if(bi_region) { //multiplier
    miou_yield=PHASECTE[ppx].Bi_multi*visco;
  }
  //Papanastasiou
  float miouPap=miou_yield *(1.f-exp(-m_NN*D_tensor_magn));
  float visco_etap1_term1=(PHASECTE[ppx].tau_max ? miou_yield : (miouPap>m_NN*tau_yield||D_tensor_magn==ALMOSTZERO ? m_NN*tau_yield : miouPap));

  //HB
  float miouHB=visco*pow(D_tensor_magn,(n_NN-1.0f));
  float visco_etap1_term2=(bi_region ? visco : (miouPap>m_NN*tau_yield||D_tensor_magn==ALMOSTZERO ? visco : miouHB));

  visco_etap1=visco_etap1_term1+visco_etap1_term2;

  /*
  //use according to you criteria
  - Herein we limit visco_etap1 at very low shear rates
  */
}

//------------------------------------------------------------------------------
/// Calclulate stress tensor.
/// Calcular tensor de estres.
//------------------------------------------------------------------------------
/*
__device__ void GetStressTensor(const tmatrix3f &D_tensor,float visco_etap1,float &I_t,float &II_t,float &J1_t
  ,float &J2_t,float &tau_tensor_magn,tmatrix3f &tau_tensor)
{
  //Stress tensor and invariant
  tau_tensor.a11=2.f*visco_etap1*(D_tensor.a11);	tau_tensor.a12=2.f*visco_etap1*D_tensor.a12;		tau_tensor.a13=2.f*visco_etap1*D_tensor.a13;
  tau_tensor.a21=2.f*visco_etap1*D_tensor.a21;		tau_tensor.a22=2.f*visco_etap1*(D_tensor.a22);	tau_tensor.a23=2.f*visco_etap1*D_tensor.a23;
  tau_tensor.a31=2.f*visco_etap1*D_tensor.a31;		tau_tensor.a32=2.f*visco_etap1*D_tensor.a32;		tau_tensor.a33=2.f*visco_etap1*(D_tensor.a33);

  //I_t - the first invariant -
  I_t=tau_tensor.a11+tau_tensor.a22+tau_tensor.a33;
  //II_t - the second invariant - expnaded form witout symetry 
  float II_t_1=tau_tensor.a11*tau_tensor.a22+tau_tensor.a22*tau_tensor.a33+tau_tensor.a11*tau_tensor.a33;
  float II_t_2=tau_tensor.a12*tau_tensor.a21+tau_tensor.a23*tau_tensor.a32+tau_tensor.a13*tau_tensor.a31;
  II_t=II_t_1-II_t_2;
  //stress tensor magnitude
  tau_tensor_magn=sqrt(II_t*II_t);

  //Main Strain rate invariants
  J1_t=I_t; J2_t=I_t*I_t-2.f*II_t;
}
*/
//##############################################################################
//# Kernels for calculating forces (Pos-Double) for non-Newtonian models.
//# Kernels para calculo de fuerzas (Pos-Double) para modelos no-Newtonianos.
//##############################################################################
//------------------------------------------------------------------------------
/// Interaction of a particle with a set of particles. Bound-Fluid/Float
/// Realiza la interaccion de una particula con un conjunto de ellas. Bound-Fluid/Float
//------------------------------------------------------------------------------
template<TpKernel tker,TpFtMode ftmode,TpVisco tvisco, bool symm>
__device__ void KerInteractionForcesBoundBox_NN
(unsigned p1,const unsigned &pini,const unsigned &pfin
  ,const float *ftomassp, float2 *tauff
  ,const float4 *poscell,const float4 *velrhop,const typecode *code,const unsigned* idp
  ,float massf,const float4 &pscellp1,const float4 &velrhop1,float &arp1,float &visc
  ,float2 &tau_sump1_xx_xy, float2 &tau_sump1_xz_yy, float2 &tau_sump1_yz_zz
  ,float4 &relative_pos_sump1, float4 &velrhop1_sum)
{
  for(int p2=pini; p2<pfin; p2++) {
    const float4 pscellp2=poscell[p2];
    float drx=pscellp1.x-pscellp2.x+CTE.poscellsize*(CEL_GetX(__float_as_int(pscellp1.w))-CEL_GetX(__float_as_int(pscellp2.w)));
    float dry=pscellp1.y-pscellp2.y+CTE.poscellsize*(CEL_GetY(__float_as_int(pscellp1.w))-CEL_GetY(__float_as_int(pscellp2.w)));
    float drz=pscellp1.z-pscellp2.z+CTE.poscellsize*(CEL_GetZ(__float_as_int(pscellp1.w))-CEL_GetZ(__float_as_int(pscellp2.w)));
    if(symm)dry=pscellp1.y+pscellp2.y+CTE.poscellsize*CEL_GetY(__float_as_int(pscellp2.w)); //<vs_syymmetry>
    const float rr2=drx*drx+dry*dry+drz*drz;
    if(rr2<=CTE.kernelsize2 && rr2>=ALMOSTZERO) {
      //-Computes kernel.
      const float fac=cufsph::GetKernel_Fac<tker>(rr2);
      const float frx=fac*drx,fry=fac*dry,frz=fac*drz; //-Gradients.

      float4 velrhop2=velrhop[p2];
      if(symm)velrhop2.y=-velrhop2.y; //<vs_syymmetry>

      float massp2 = massf;
      const typecode pp2=CODE_GetTypeValue(code[p2]);
      if(tvisco==VISCO_Hypoplasticity){
        massp2=PHASEHYPO[pp2].mass;
      }
      if(tvisco==VISCO_Elasticity){
        massp2=PHASEELASTIC[pp2].mass;
      }

      //-Obtains particle mass p2 if there are floating bodies.
      //-Obtiene masa de particula p2 en caso de existir floatings.
      float ftmassp2;    //-Contains mass of floating body or massf if fluid. | Contiene masa de particula floating o massf si es fluid.
      bool compute=true; //-Deactivated when DEM is used and is float-float or float-bound. | Se desactiva cuando se usa DEM y es float-float o float-bound.
      if(USE_FLOATING) {
        const typecode cod=code[p2];
        bool ftp2=CODE_IsFloating(cod);
        ftmassp2=(ftp2 ? ftomassp[CODE_GetTypeValue(cod)] : massp2);
        compute=!(USE_FTEXTERNAL && ftp2); //-Deactivated when DEM or Chrono is used and is bound-float. | Se desactiva cuando se usa DEM o Chrono y es bound-float.
      }

      if(compute) {
        //-Density derivative (Continuity equation).
        const float dvx=velrhop1.x-velrhop2.x,dvy=velrhop1.y-velrhop2.y,dvz=velrhop1.z-velrhop2.z;
        arp1+=(USE_FLOATING ? ftmassp2 : massp2)*(dvx*frx+dvy*fry+dvz*frz)*(velrhop1.w/velrhop2.w);

        if(tvisco!=VISCO_Artificial) {	
          float2 taup2_xx_xy=tauff[p2*3];
          float2 taup2_xz_yy=tauff[p2*3+1];
          float2 taup2_yz_zz=tauff[p2*3+2];
          const float vol_ker = fac*massp2/velrhop2.w;
          tau_sump1_xx_xy.x += taup2_xx_xy.x*vol_ker;
          tau_sump1_xx_xy.y += taup2_xx_xy.y*vol_ker;
          tau_sump1_xz_yy.x += taup2_xz_yy.x*vol_ker;
          tau_sump1_xz_yy.y += taup2_xz_yy.y*vol_ker;
          tau_sump1_yz_zz.x += taup2_yz_zz.x*vol_ker;
          tau_sump1_yz_zz.y += taup2_yz_zz.y*vol_ker;

          velrhop1_sum.x +=  velrhop2.x*vol_ker;
          velrhop1_sum.y +=  velrhop2.y*vol_ker;
          velrhop1_sum.z +=  velrhop2.z*vol_ker;

          relative_pos_sump1.x += drx*velrhop2.w*vol_ker;
          relative_pos_sump1.y += dry*velrhop2.w*vol_ker;
          relative_pos_sump1.z += drz*velrhop2.w*vol_ker;
          relative_pos_sump1.w += vol_ker;
        }

        {//===== Viscosity ===== 
          const float dot=drx*dvx+dry*dvy+drz*dvz;
          const float dot_rr2=dot/(rr2+CTE.eta2);
          visc=max(dot_rr2,visc);
        }
      }
    }
  }
}

//------------------------------------------------------------------------------
/// Particle interaction for non-Newtonian models. Bound-Fluid/Float 
/// Realiza interaccion entre particulas para modelos no-Newtonianos. Bound-Fluid/Float
//------------------------------------------------------------------------------
template<TpKernel tker,TpFtMode ftmode, TpVisco tvisco, bool symm>
__global__ void KerInteractionForcesBound_NN(unsigned n,unsigned pinit
  ,int scelldiv,int4 nc,int3 cellzero,const int2 *beginendcellfluid,const unsigned *dcell
  ,const float *ftomassp, float2 *tauff
  ,const float4 *poscell, float4 *velrhop,const typecode *code,const unsigned *idp
  ,float *viscdt,float *ar, float3 *ace, double dt)
{
  const unsigned p=blockIdx.x*blockDim.x+threadIdx.x; //-Number of thread.
  if(p<n) {
    const unsigned p1=p+pinit;      //-Number of particle.
    float visc=0,arp1=0;

    //initialize variables for integration
    float4 relative_pos_sump1 = make_float4(0,0,0,0);
    float4 velrhop1_sum = make_float4(0,0,0,0);
    float2 tau_sump1_xx_xy = make_float2(0,0);
    float2 tau_sump1_xz_yy = make_float2(0,0);
    float2 tau_sump1_yz_zz = make_float2(0,0);

    //-Loads particle p1 data.
    const float4 pscellp1=poscell[p1];
    const float4 velrhop1=velrhop[p1];
    const float3 acep1=ace[p1];
    const bool rsymp1=(symm && CEL_GetPartY(__float_as_uint(pscellp1.w))==0); //<vs_syymmetry>

    //-Obtains neighborhood search limits.
    int ini1,fin1,ini2,fin2,ini3,fin3;
    cunsearch::InitCte(dcell[p1],scelldiv,nc,cellzero,ini1,fin1,ini2,fin2,ini3,fin3);

    //jinw@need check the fluid/granular particle mass is correct or not
    //-Boundary-Fluid interaction.
    for(int c3=ini3; c3<fin3; c3+=nc.w)for(int c2=ini2; c2<fin2; c2+=nc.x) {
      unsigned pini,pfin=0; cunsearch::ParticleRange(c2,c3,ini1,fin1,beginendcellfluid,pini,pfin);
      if(pfin) {
        KerInteractionForcesBoundBox_NN<tker,ftmode,tvisco,false>(p1,pini,pfin,ftomassp,tauff,poscell,velrhop,code,idp,CTE.massf,pscellp1,velrhop1,arp1,visc,tau_sump1_xx_xy,tau_sump1_xz_yy,tau_sump1_yz_zz,relative_pos_sump1,velrhop1_sum);
        if(symm && rsymp1)KerInteractionForcesBoundBox_NN<tker,ftmode,tvisco,true >(p1,pini,pfin,ftomassp,tauff,poscell,velrhop,code,idp,CTE.massf,pscellp1,velrhop1,arp1,visc,tau_sump1_xx_xy,tau_sump1_xz_yy,tau_sump1_yz_zz,relative_pos_sump1,velrhop1_sum);
      }
    }
    //-Stores results.
    if(arp1||visc) {
      ar[p1]+=arp1;
      if(visc>viscdt[p1])viscdt[p1]=visc;
    }
    if ((tvisco == VISCO_Hypoplasticity) & (relative_pos_sump1.w != 0) ){
      const float3 Gravity= CTE.gravity;
      tauff[p1*3].x   = (tau_sump1_xx_xy.x - (Gravity.x-velrhop1.x/dt)*relative_pos_sump1.x)/relative_pos_sump1.w;
      tauff[p1*3].y   = tau_sump1_xx_xy.y /relative_pos_sump1.w;
      tauff[p1*3+1].x = tau_sump1_xz_yy.x /relative_pos_sump1.w;
      tauff[p1*3+1].y = (tau_sump1_xz_yy.y - (Gravity.y-velrhop1.y/dt)*relative_pos_sump1.y)/relative_pos_sump1.w;
      tauff[p1*3+2].x = tau_sump1_yz_zz.x /relative_pos_sump1.w;
      tauff[p1*3+2].y = (tau_sump1_yz_zz.y - (Gravity.z-velrhop1.z/dt)*relative_pos_sump1.z)/relative_pos_sump1.w;

      velrhop[p1].x = 2*velrhop1.x - velrhop1_sum.x/relative_pos_sump1.w;
      velrhop[p1].y = 2*velrhop1.y - velrhop1_sum.y/relative_pos_sump1.w;
      velrhop[p1].z = 2*velrhop1.z - velrhop1_sum.z/relative_pos_sump1.w;
    }
  }
}
//======================Start of FDA approach===================================
//------------------------------------------------------------------------------
/// Interaction of a particle with a set of particles for non-Newtonian models using the FDA approach. (Fluid/Float-Fluid/Float/Bound)
/// Realiza la interaccion de una particula con un conjunto de ellas para modelos no Newtonianos que utilizan el enfoque de la FDA. (Fluid/Float-Fluid/Float/Bound)
//------------------------------------------------------------------------------
/*
template<TpKernel tker,TpFtMode ftmode,TpVisco tvisco,TpDensity tdensity,bool shift,bool symm>
__device__ void KerInteractionForcesFluidBox_FDA(bool boundp2,unsigned p1
  ,const unsigned &pini,const unsigned &pfin,float visco,float *visco_eta
  ,const float *ftomassp,float2 *tauff
  ,const float4 *poscell,const float4 *velrhop,const typecode *code,const unsigned *idp
  ,float massp2,const typecode pp1,bool ftp1
  ,const float4 &pscellp1,const float4 &velrhop1,float pressp1
  ,float2 &taup1_xx_xy,float2 &taup1_xz_yy,float2 &taup1_yz_zz
  ,float3 &grap1_xx_xy_xz,float3 &grap1_yx_yy_yz,float3 &grap1_zx_zy_zz
  ,float3 &acep1,float &arp1,float &visc,float &visceta,float &visco_etap1,float &deltap1
  ,TpShifting shiftmode,float4 &shiftposfsp1)
{
  for(int p2=pini; p2<pfin; p2++){
    const float4 pscellp2=poscell[p2];
    float drx=pscellp1.x-pscellp2.x+CTE.poscellsize*(CEL_GetX(__float_as_int(pscellp1.w))-CEL_GetX(__float_as_int(pscellp2.w)));
    float dry=pscellp1.y-pscellp2.y+CTE.poscellsize*(CEL_GetY(__float_as_int(pscellp1.w))-CEL_GetY(__float_as_int(pscellp2.w)));
    float drz=pscellp1.z-pscellp2.z+CTE.poscellsize*(CEL_GetZ(__float_as_int(pscellp1.w))-CEL_GetZ(__float_as_int(pscellp2.w)));
    if(symm)dry=pscellp1.y+pscellp2.y+CTE.poscellsize*CEL_GetY(__float_as_int(pscellp2.w)); //<vs_syymmetry>
    const float rr2=drx*drx+dry*dry+drz*drz;
    if(rr2<=CTE.kernelsize2 && rr2>=ALMOSTZERO){
      //-Computes kernel.
      const float fac=cufsph::GetKernel_Fac<tker>(rr2);
      const float frx=fac*drx,fry=fac*dry,frz=fac*drz; //-Gradients.

      //-Obtains mass of particle p2 for NN and if any floating bodies exist.
      const typecode cod=code[p2];
      const typecode pp2=(boundp2 ? pp1 : CODE_GetTypeValue(cod)); //<vs_non-Newtonian>
      float massp2=(boundp2 ? CTE.massb : PHASEARRAY[pp2].mass); //massp2 not neccesary to go in _Box function
      //Note if you masses are very different more than a ratio of 1.3 then: massp2 = (boundp2 ? PHASEARRAY[pp1].mass : PHASEARRAY[pp2].mass);

      //-Obtiene masa de particula p2 en caso de existir floatings.
      bool ftp2=false;         //-Indicates if it is floating. | Indica si es floating.
      float ftmassp2;    //-Contains mass of floating body or massf if fluid. | Contiene masa de particula floating o massp2 si es bound o fluid.
      bool compute=true; //-Deactivated when DEM is used and is float-float or float-bound. | Se desactiva cuando se usa DEM y es float-float o float-bound.
      if(USE_FLOATING) {
        ftp2=CODE_IsFloating(cod);
        ftmassp2=(ftp2 ? ftomassp[CODE_GetTypeValue(cod)] : massp2);
#ifdef DELTA_HEAVYFLOATING
        if(ftp2 && tdensity==DDT_DDT && ftmassp2<=(massp2*1.2f))deltap1=FLT_MAX;
#else
        if(ftp2 && tdensity==DDT_DDT)deltap1=FLT_MAX;
#endif
        if(ftp2 && shift && shiftmode==SHIFT_NoBound)shiftposfsp1.x=FLT_MAX; //-Cancels shifting with floating bodies. | Con floatings anula shifting.
        compute=!(USE_FTEXTERNAL && ftp1&&(boundp2||ftp2)); //-Deactivated when DEM or Chrono is used and is float-float or float-bound. | Se desactiva cuando se usa DEM o Chrono y es float-float o float-bound.
      }

      float4 velrhop2=velrhop[p2];
      if(symm)velrhop2.y=-velrhop2.y; //<vs_syymmetry>

      //===== Aceleration ===== 
      if(compute) {
        const float pressp2=cufsph::ComputePressCte_NN(velrhop2.w,PHASEARRAY[pp2].rho,PHASEARRAY[pp2].CteB,PHASEARRAY[pp2].Gamma,PHASEARRAY[pp2].Cs0,cod);
        const float prs=(pressp1+pressp2)/(velrhop1.w*velrhop2.w)
          +(tker==KERNEL_Cubic ? cufsph::GetKernelCubic_Tensil(rr2,velrhop1.w,pressp1,velrhop2.w,pressp2) : 0);
        const float p_vpm=-prs*(USE_FLOATING ? ftmassp2 : massp2);
        acep1.x+=p_vpm*frx; acep1.y+=p_vpm*fry; acep1.z+=p_vpm*frz;
      }

      //-Density derivative (Continuity equation).
      float dvx=velrhop1.x-velrhop2.x,dvy=velrhop1.y-velrhop2.y,dvz=velrhop1.z-velrhop2.z;
      if(compute)arp1+=(USE_FLOATING ? ftmassp2 : massp2)*(dvx*frx+dvy*fry+dvz*frz)*(velrhop1.w/velrhop2.w);

      const float cbar=max(PHASEARRAY[pp2].Cs0,PHASEARRAY[pp2].Cs0);
      const float dot3=(tdensity!=DDT_None||shift ? drx*frx+dry*fry+drz*frz : 0);
      //-Density derivative (DeltaSPH Molteni).
      if(tdensity==DDT_DDT && deltap1!=FLT_MAX) {
        const float rhop1over2=velrhop1.w/velrhop2.w;
        const float visc_densi=CTE.ddtkh*cbar*(rhop1over2-1.f)/(rr2+CTE.eta2);
        const float delta=(pp1==pp2 ? visc_densi*dot3*(USE_FLOATING ? ftmassp2 : massp2) : 0); //<vs_non-Newtonian>
        //deltap1=(boundp2? FLT_MAX: deltap1+delta);
        deltap1=(boundp2 && CTE.tboundary==BC_DBC ? FLT_MAX : deltap1+delta);
      }
      //-Density Diffusion Term (Fourtakas et al 2019). //<vs_dtt2_ini>
      if((tdensity==DDT_DDT2||(tdensity==DDT_DDT2Full&&!boundp2))&&deltap1!=FLT_MAX&&!ftp2) {
        const float rh=1.f+CTE.ddtgz*drz;
        const float drhop=CTE.rhopzero*pow(rh,1.f/CTE.gamma)-CTE.rhopzero;
        const float visc_densi=CTE.ddtkh*cbar*((velrhop2.w-velrhop1.w)-drhop)/(rr2+CTE.eta2);
        const float delta=(pp1==pp2 ? visc_densi*dot3*massp2/velrhop2.w : 0); //<vs_non-Newtonian>
        deltap1=(boundp2 ? FLT_MAX : deltap1-delta);
      } //<vs_dtt2_end>		

      //-Shifting correction.
      if(shift && shiftposfsp1.x!=FLT_MAX) {
        bool heavyphase=(PHASEARRAY[pp1].mass>PHASEARRAY[pp2].mass && pp1!=pp2 ? true : false); //<vs_non-Newtonian>
        const float massrhop=(USE_FLOATING ? ftmassp2 : massp2)/velrhop2.w;
        const bool noshift=(boundp2&&(shiftmode==SHIFT_NoBound||(shiftmode==SHIFT_NoFixed && CODE_IsFixed(code[p2]))));
        shiftposfsp1.x=(noshift ? FLT_MAX : (heavyphase ? 0 : shiftposfsp1.x+massrhop*frx)); //-Removes shifting for the boundaries. | Con boundary anula shifting.
        shiftposfsp1.y+=(heavyphase ? 0 : massrhop*fry);
        shiftposfsp1.z+=(heavyphase ? 0 : massrhop*frz);
        shiftposfsp1.w-=(heavyphase ? 0 : massrhop*dot3);
      }

      //===== Viscosity ===== 
      if(compute) {
        const float dot=drx*dvx+dry*dvy+drz*dvz;
        const float dot_rr2=dot/(rr2+CTE.eta2);
        visc=max(dot_rr2,visc);  //ViscDt=max(dot/(rr2+Eta2),ViscDt);
        //<vs_non-Newtonian>
        const float visco_NN=PHASECTE[pp2].visco;
        if(tvisco==VISCO_Artificial) {//-Artificial viscosity.
          if(dot<0) {
            const float amubar=CTE.kernelh*dot_rr2;  //amubar=CTE.kernelh*dot/(rr2+CTE.eta2);
            const float robar=(velrhop1.w+velrhop2.w)*0.5f;
            const float pi_visc=(-visco_NN*cbar*amubar/robar)*(USE_FLOATING ? ftmassp2 : massp2);
            acep1.x-=pi_visc*frx; acep1.y-=pi_visc*fry; acep1.z-=pi_visc*frz;
          }
        }
        else if(tvisco==VISCO_LaminarSPS||tvisco==VISCO_ConstEq) {
          {
            //vel gradients
            if(boundp2) { //this applies no slip on stress tensor
              dvx=2.f*velrhop1.x; dvy=2.f*velrhop1.y; dvz=2.f*velrhop1.z;  //fomraly I should use the moving BC vel as ug=2ub-uf
            }
            tmatrix3f dvelp1; float div_vel;
            GetVelocityGradients_FDA(rr2,drx,dry,drz,dvx,dvy,dvz,dvelp1,div_vel);

            //Strain rate tensor 
            tmatrix3f D_tensor; float div_D_tensor; float D_tensor_magn;
            float I_D,II_D; float J1_D,J2_D;
            GetStrainRateTensor(dvelp1,div_vel,I_D,II_D,J1_D,J2_D,div_D_tensor,D_tensor_magn,D_tensor);

            //Effective viscosity
            float m_NN=PHASECTE[pp2].m_NN; float n_NN=PHASECTE[pp2].n_NN; float tau_yield=PHASECTE[pp2].tau_yield;
            KerGetEta_Effective(pp1,tau_yield,D_tensor_magn,visco_NN,m_NN,n_NN,visco_etap1);
            visceta=max(visceta,visco_etap1);

            if(tvisco==VISCO_LaminarSPS){ //-Laminar contribution.
              //Morris Operator
              const float temp=2.f*(visco_etap1)/((rr2+CTE.eta2)*velrhop2.w);  //-Note this is the Morris operator and not Lo and Shao
              const float vtemp=(USE_FLOATING ? ftmassp2 : massp2)*temp*(drx*frx+dry*fry+drz*frz);
              acep1.x+=vtemp*dvx; acep1.y+=vtemp*dvy; acep1.z+=vtemp*dvz;

            }
            else if(tvisco==VISCO_ConstEq) {
              //stress tensor tau 
              tmatrix3f tau_tensor; float tau_tensor_magn;
              float I_t,II_t; float J1_t,J2_t;
              GetStressTensor(D_tensor,visco_etap1,I_t,II_t,J1_t,J2_t,tau_tensor_magn,tau_tensor);

              //viscous forces
              float taux=(tau_tensor.a11*frx+tau_tensor.a12*fry+tau_tensor.a13*frz)/(velrhop2.w); //Morris 1997
              float tauy=(tau_tensor.a21*frx+tau_tensor.a22*fry+tau_tensor.a23*frz)/(velrhop2.w);
              float tauz=(tau_tensor.a31*frx+tau_tensor.a32*fry+tau_tensor.a33*frz)/(velrhop2.w);
              const float mtemp=(USE_FLOATING ? ftmassp2 : massp2);
              acep1.x+=taux*mtemp; acep1.y+=tauy*mtemp; acep1.z+=tauz*mtemp;
            }
          }
          //-SPS turbulence model.
          //-SPS turbulence model is disabled in v5.0 NN version
        }
      }
    }
  }
}

//------------------------------------------------------------------------------
/// Interaction between particles for non-Newtonian models using the FDA approach. Fluid/Float-Fluid/Float or Fluid/Float-Bound.
/// Includes artificial/laminar/Const Eq. viscosity and normal/DEM floating bodies.
///
/// Realiza interaccion entre particulas para modelos no-Newtonianos que utilizan el enfoque de la FDA. Fluid/Float-Fluid/Float or Fluid/Float-Bound
/// Incluye visco artificial/laminar y floatings normales/dem.
//------------------------------------------------------------------------------
template<TpKernel tker,TpFtMode ftmode,TpVisco tvisco,TpDensity tdensity,bool shift,bool symm>
__global__ void KerInteractionForcesFluid_NN_FDA(unsigned n,unsigned pinit,float viscob,float viscof,float *visco_eta
  ,int scelldiv,int4 nc,int3 cellzero,const int2 *begincell,unsigned cellfluid,const unsigned *dcell
  ,const float *ftomassp,float2 *tauff,float3 *gradvelff
  ,const float4 *poscell,const float4 *velrhop
  ,const typecode *code,const unsigned *idp
  ,float *viscdt,float *viscetadt,float *ar,float3 *ace,float *delta
  ,TpShifting shiftmode,float4 *shiftposfs)
{
  const unsigned p=blockIdx.x*blockDim.x+threadIdx.x; //-Number of particle.
  if(p<n) {
    unsigned p1=p+pinit;      //-Number of particle.
    float visc=0,arp1=0,deltap1=0;
    float3 acep1=make_float3(0,0,0);

    //-Variables for Shifting.
    float4 shiftposfsp1;
    if(shift)shiftposfsp1=shiftposfs[p1];

    //-Obtains data of particle p1 in case there are floating bodies.		
    bool ftp1;       //-Indicates if it is floating. | Indica si es floating.
    const typecode cod=code[p1];
    if(USE_FLOATING) {
      ftp1=CODE_IsFloating(cod);
      if(ftp1 && tdensity!=DDT_None)deltap1=FLT_MAX; //-DDT is not applied to floating particles.
      if(ftp1 && shift)shiftposfsp1.x=FLT_MAX; //-Shifting is not calculated for floating bodies. | Para floatings no se calcula shifting.
    }

    //-Obtains basic data of particle p1.
    const float4 pscellp1=poscell[p1];
    const float4 velrhop1=velrhop[p1];
    //<vs_non-Newtonian>
    const typecode pp1=CODE_GetTypeValue(cod);
    float visco_etap1=0;
    float visceta=0;

    //Obtain pressure		
    const float pressp1=cufsph::ComputePressCte_NN(velrhop1.w,PHASEARRAY[pp1].rho,PHASEARRAY[pp1].CteB,PHASEARRAY[pp1].Gamma,PHASEARRAY[pp1].Cs0,cod);
    const bool rsymp1=(symm && CEL_GetPartY(__float_as_uint(pscellp1.w))==0); //<vs_syymmetry>

    //-Variables for Laminar+SPS.
    float2 taup1_xx_xy,taup1_xz_yy,taup1_yz_zz;
    if(tvisco!=VISCO_Artificial) {
      taup1_xx_xy=tauff[p1*3];
      taup1_xz_yy=tauff[p1*3+1];
      taup1_yz_zz=tauff[p1*3+2];
    }
    //-Variables for Laminar+SPS (computation).
    float3 grap1_xx_xy_xz,grap1_yx_yy_yz,grap1_zx_zy_zz;
    if(tvisco!=VISCO_Artificial) {
      grap1_xx_xy_xz=make_float3(0,0,0);
      grap1_yx_yy_yz=make_float3(0,0,0);
      grap1_zx_zy_zz=make_float3(0,0,0);
    }

    //-Obtains neighborhood search limits.
    int ini1,fin1,ini2,fin2,ini3,fin3;
    cunsearch::InitCte(dcell[p1],scelldiv,nc,cellzero,ini1,fin1,ini2,fin2,ini3,fin3);

    //-Interaction with fluids.
    ini3+=cellfluid; fin3+=cellfluid;
    for(int c3=ini3; c3<fin3; c3+=nc.w)for(int c2=ini2; c2<fin2; c2+=nc.x){
      unsigned pini,pfin=0;  cunsearch::ParticleRange(c2,c3,ini1,fin1,begincell,pini,pfin);
      if(pfin){
        KerInteractionForcesFluidBox_FDA<tker,ftmode,tvisco,tdensity,shift,false>(false,p1,pini,pfin,viscof,visco_eta,ftomassp,tauff,poscell,velrhop,code,idp,CTE.massf,pp1,ftp1,pscellp1,velrhop1,pressp1,taup1_xx_xy,taup1_xz_yy,taup1_yz_zz,grap1_xx_xy_xz,grap1_yx_yy_yz,grap1_zx_zy_zz,acep1,arp1,visc,visceta,visco_etap1,deltap1,shiftmode,shiftposfsp1);
        if(symm && rsymp1)	KerInteractionForcesFluidBox_FDA<tker,ftmode,tvisco,tdensity,shift,true >(false,p1,pini,pfin,viscof,visco_eta,ftomassp,tauff,poscell,velrhop,code,idp,CTE.massf,pp1,ftp1,pscellp1,velrhop1,pressp1,taup1_xx_xy,taup1_xz_yy,taup1_yz_zz,grap1_xx_xy_xz,grap1_yx_yy_yz,grap1_zx_zy_zz,acep1,arp1,visc,visceta,visco_etap1,deltap1,shiftmode,shiftposfsp1); //<vs_syymmetry>
      }
    }
    //-Interaction with boundaries.
    ini3-=cellfluid; fin3-=cellfluid;
    for(int c3=ini3; c3<fin3; c3+=nc.w)for(int c2=ini2; c2<fin2; c2+=nc.x){
      unsigned pini,pfin=0;  cunsearch::ParticleRange(c2,c3,ini1,fin1,begincell,pini,pfin);
      if(pfin){
        KerInteractionForcesFluidBox_FDA<tker,ftmode,tvisco,tdensity,shift,false>(true,p1,pini,pfin,viscob,visco_eta,ftomassp,tauff,poscell,velrhop,code,idp,CTE.massf,pp1,ftp1,pscellp1,velrhop1,pressp1,taup1_xx_xy,taup1_xz_yy,taup1_yz_zz,grap1_xx_xy_xz,grap1_yx_yy_yz,grap1_zx_zy_zz,acep1,arp1,visc,visceta,visco_etap1,deltap1,shiftmode,shiftposfsp1);
        if(symm && rsymp1)	KerInteractionForcesFluidBox_FDA<tker,ftmode,tvisco,tdensity,shift,true >(true,p1,pini,pfin,viscob,visco_eta,ftomassp,tauff,poscell,velrhop,code,idp,CTE.massf,pp1,ftp1,pscellp1,velrhop1,pressp1,taup1_xx_xy,taup1_xz_yy,taup1_yz_zz,grap1_xx_xy_xz,grap1_yx_yy_yz,grap1_zx_zy_zz,acep1,arp1,visc,visceta,visco_etap1,deltap1,shiftmode,shiftposfsp1);  //<vs_syymmetry>
      }
    }
    //-Stores results.
    if(shift||arp1||acep1.x||acep1.y||acep1.z||visc||visceta||visco_etap1) {
      if(tdensity!=DDT_None) {
        if(delta) {
          const float rdelta=delta[p1];
          delta[p1]=(rdelta==FLT_MAX||deltap1==FLT_MAX ? FLT_MAX : rdelta+deltap1);
        }
        else if(deltap1!=FLT_MAX)arp1+=deltap1;
      }
      ar[p1]+=arp1;
      float3 r=ace[p1]; r.x+=acep1.x; r.y+=acep1.y; r.z+=acep1.z; ace[p1]=r;
      if(visc>viscdt[p1])viscdt[p1]=visc;
      if(visceta>viscetadt[p1])viscetadt[p1]=visceta;
      if(tvisco==VISCO_LaminarSPS) {
        float3 rg;
        rg=gradvelff[p1*3];		 rg=make_float3(rg.x+grap1_xx_xy_xz.x, rg.y+grap1_xx_xy_xz.y, rg.z+grap1_xx_xy_xz.z);  gradvelff[p1*3]=rg;
        rg=gradvelff[p1*3+1];  rg=make_float3(rg.x+grap1_yx_yy_yz.x, rg.y+grap1_yx_yy_yz.y, rg.z+grap1_yx_yy_yz.z);  gradvelff[p1*3+1]=rg;
        rg=gradvelff[p1*3+2];  rg=make_float3(rg.x+grap1_zx_zy_zz.x, rg.y+grap1_zx_zy_zz.y, rg.z+grap1_zx_zy_zz.z);  gradvelff[p1*3+2]=rg;
      }
      if(shift)shiftposfs[p1]=shiftposfsp1;
      //auxnn[p1] = visco_etap1; //to be used if an auxilary is needed for debug or otherwise.
    }
  }
}

//==============================================================================
/// Interaction for the force computation for non-Newtonian models using the FDA approach.
/// Interaccion para el calculo de fuerzas para modelos no-Newtonianos que utilizan el enfoque de la FDA.
//==============================================================================
template<TpKernel tker,TpFtMode ftmode,TpVisco tvisco,TpDensity tdensity,bool shift>
void Interaction_ForcesGpuT_NN_FDA(const StInterParmsg &t, int &Zhan_loop)
{
  //-Collects kernel information.
#ifndef DISABLE_BSMODES
  if(t.kerinfo) {
    cusph::Interaction_ForcesT_KerInfo<tker,ftmode,true,tdensity,shift,false>(t.kerinfo);
    return;
  }
#endif
  const StDivDataGpu &dvd=t.divdatag;
  const int2* beginendcell=dvd.beginendcell;
  //-Interaction Fluid-Fluid & Fluid-Bound.
  if(t.fluidnum) {
    dim3 sgridf=GetSimpleGridSize(t.fluidnum,t.bsfluid);
    if(t.symmetry) //<vs_syymmetry_ini>
      KerInteractionForcesFluid_NN_FDA<tker,ftmode,tvisco,tdensity,shift,true ><<<sgridf,t.bsfluid,0,t.stm>>>
      (t.fluidnum,t.fluidini,t.viscob,t.viscof,t.visco_eta,dvd.scelldiv,dvd.nc,dvd.cellzero,dvd.beginendcell,dvd.cellfluid,t.dcell
        ,t.ftomassp,(float2*)t.tau,(float3*)t.gradvel,t.poscell,t.velrhop,t.code,t.idp
        ,t.viscdt,t.viscetadt,t.ar,t.ace,t.delta,t.shiftmode,t.shiftposfs);
    else //<vs_syymmetry_end>
      KerInteractionForcesFluid_NN_FDA<tker,ftmode,tvisco,tdensity,shift,false><<<sgridf,t.bsfluid,0,t.stm>>>
      (t.fluidnum,t.fluidini,t.viscob,t.viscof,t.visco_eta,dvd.scelldiv,dvd.nc,dvd.cellzero,dvd.beginendcell,dvd.cellfluid,t.dcell
        ,t.ftomassp,(float2*)t.tau,(float3*)t.gradvel,t.poscell,t.velrhop,t.code,t.idp
        ,t.viscdt,t.viscetadt,t.ar,t.ace,t.delta,t.shiftmode,t.shiftposfs);
  }
  //-Interaction Boundary-Fluid.
  if(t.boundnum) {
    const int2* beginendcellfluid=dvd.beginendcell+dvd.cellfluid;
    dim3 sgridb=GetSimpleGridSize(t.boundnum,t.bsbound);
    //printf("bsbound:%u\n",bsbound);
    if(t.symmetry) //<vs_syymmetry_ini>
      KerInteractionForcesBound_NN<tker,ftmode,tvisco,true ><<<sgridb,t.bsbound,0,t.stm>>>
      (t.boundnum,t.boundini,dvd.scelldiv,dvd.nc,dvd.cellzero,beginendcellfluid,t.dcell
        ,t.ftomassp,(float2*)t.tau,t.poscell,t.velrhop,t.code,t.idp,t.viscdt,t.ar,t.ace,time_inc);
    else //<vs_syymmetry_end>
      KerInteractionForcesBound_NN<tker,ftmode,tvisco,false><<<sgridb,t.bsbound,0,t.stm>>>
      (t.boundnum,t.boundini,dvd.scelldiv,dvd.nc,dvd.cellzero,beginendcellfluid,t.dcell
        ,t.ftomassp,(float2*)t.tau,t.poscell,t.velrhop,t.code,t.idp,t.viscdt,t.ar,t.ace,time_inc);
  }
}
*/
//======================END of FDA==============================================

//======================Start of SPH============================================
//------------------------------------------------------------------------------
/// Interaction of a particle with a set of particles for non-Newtonian models using the SPH approach with Const Eq. (Fluid/Float-Fluid/Float/Bound)
/// Realiza la interaccion de una particula con un conjunto de ellas para modelos no-Newtonianos que utilizan el enfoque de la SPH Const. Eq. (Fluid/Float-Fluid/Float/Bound)
//------------------------------------------------------------------------------

template<TpKernel tker,TpFtMode ftmode,TpVisco tvisco,bool symm>
__device__ void KerInteractionForcesFluidBox_SPH_ConsEq(bool boundp2,const unsigned &pini,const unsigned &pfin
  ,const float *ftomassp,float2 *tauff,const float4 *poscell,const float4 *velrhop,const typecode *code
  ,const typecode pp1,bool ftp1,const float4 &pscellp1,const float4 &velrhop1,float2 &taup1_xx_xy,float2 &taup1_xz_yy,float2 &taup1_yz_zz
  ,float3 &acep1,float &visc,float3 &taup1_yy_yz_zz, unsigned p1,float4 & taup1_xx_xy_xz_sum,bool regularize)
{
  for(int p2=pini; p2<pfin; p2++) {
    const float4 pscellp2=poscell[p2];
    float drx=pscellp1.x-pscellp2.x+CTE.poscellsize*(CEL_GetX(__float_as_int(pscellp1.w))-CEL_GetX(__float_as_int(pscellp2.w)));
    float dry=pscellp1.y-pscellp2.y+CTE.poscellsize*(CEL_GetY(__float_as_int(pscellp1.w))-CEL_GetY(__float_as_int(pscellp2.w)));
    float drz=pscellp1.z-pscellp2.z+CTE.poscellsize*(CEL_GetZ(__float_as_int(pscellp1.w))-CEL_GetZ(__float_as_int(pscellp2.w)));
    if(symm)dry=pscellp1.y+pscellp2.y+CTE.poscellsize*CEL_GetY(__float_as_int(pscellp2.w)); //<vs_syymmetry>
    const float rr2=drx*drx+dry*dry+drz*drz;
    if(rr2<=CTE.kernelsize2 && rr2>=ALMOSTZERO) {
      //-Computes kernel.
      const float fac=cufsph::GetKernel_Fac<tker>(rr2);
      const float frx=fac*drx,fry=fac*dry,frz=fac*drz; //-Gradients.

      //-Obtains mass of particle p2 for NN and if any floating bodies exist.
      const typecode cod=code[p2];
      const typecode pp2=(boundp2 ? pp1 : CODE_GetTypeValue(cod)); //<vs_non-Newtonian>
      float massp2;
      if(tvisco==VISCO_Hypoplasticity){
         massp2=(boundp2 ? CTE.massb : PHASEHYPO[pp2].mass); 
      }else if(tvisco==VISCO_Elasticity){
         massp2=(boundp2 ? CTE.massb : PHASEELASTIC[pp2].mass);
      }else{
         massp2=(boundp2 ? CTE.massb : PHASEARRAY[pp2].mass); 
      }
      //Note if you masses are very different more than a ratio of 1.3 then: massp2 = (boundp2 ? PHASEARRAY[pp1].mass : PHASEARRAY[pp2].mass);

      //-Obtiene masa de particula p2 en caso de existir floatings.
      bool ftp2=false;         //-Indicates if it is floating. | Indica si es floating.
      float ftmassp2;    //-Contains mass of floating body or massf if fluid. | Contiene masa de particula floating o massp2 si es bound o fluid.
      bool compute=true; //-Deactivated when DEM is used and is float-float or float-bound. | Se desactiva cuando se usa DEM y es float-float o float-bound.
      if(USE_FLOATING) {
        const typecode cod=code[p2];
        ftp2=CODE_IsFloating(cod);
        ftmassp2=(ftp2 ? ftomassp[CODE_GetTypeValue(cod)] : massp2);
        compute=!(USE_FTEXTERNAL && ftp1&&(boundp2||ftp2)); //-Deactivated when DEM or Chrono is used and is float-float or float-bound. | Se desactiva cuando se usa DEM o Chrono y es float-float o float-bound.
      }

      float4 velrhop2=velrhop[p2];
      if(symm)velrhop2.y=-velrhop2.y; //<vs_syymmetry>

      //-velocity dvx.
      const float dvx=velrhop1.x-velrhop2.x,dvy=velrhop1.y-velrhop2.y,dvz=velrhop1.z-velrhop2.z;
      //===== Viscosity ===== 
      if(compute) {
        const float dot=drx*dvx+dry*dvy+drz*dvz;
        const float dot_rr2=dot/(rr2+CTE.eta2);
        visc=max(dot_rr2,visc);  //ViscDt=max(dot/(rr2+Eta2),ViscDt);

        //<vs_non-Newtonian>				
        float2 tau_sum_xx_xy=make_float2(0,0);
        float2 tau_sum_xz_yy=make_float2(0,0);
        float2 tau_sum_yz_zz=make_float2(0,0);
        float2 taup2_xx_xy=tauff[p2*3];
        float2 taup2_xz_yy=tauff[p2*3+1];
        float2 taup2_yz_zz=tauff[p2*3+2];

        if(boundp2 & (tvisco != VISCO_Hypoplasticity) & (tvisco != VISCO_Elasticity)){ 
          // taup2_xx_xy=taup1_xx_xy; // use (-) for slip and (+1) for no slip
          // taup2_xz_yy=taup1_xz_yy;
          // taup2_yz_zz=taup1_yz_zz;
          float MaxSigma= min(taup1_xx_xy.x, min(taup1_xz_yy.y,taup1_yz_zz.y));
          taup2_xx_xy=make_float2(MaxSigma,taup1_xx_xy.y); 
          taup2_xz_yy=make_float2(taup1_xz_yy.x,MaxSigma);
          taup2_yz_zz=make_float2(taup1_yz_zz.x,MaxSigma);
          // velrhop2.w = velrhop1.w;
        }
        //if(regularize){
        //  taup1_xx_xy_xz_sum.x += taup2_xx_xy.x*fac;
        //  taup1_xx_xy_xz_sum.y += taup2_xx_xy.y*fac;
        //  taup1_xx_xy_xz_sum.z += taup2_xz_yy.x*fac;
        //  taup1_xx_xy_xz_sum.w += fac;
        //  taup1_yy_yz_zz.x += taup2_xz_yy.y*fac;
        //  taup1_yy_yz_zz.y += taup2_yz_zz.x*fac;
        // taup1_yy_yz_zz.z += taup2_yz_zz.y*fac;
        //}

        if(tvisco==VISCO_Hypoplasticity || tvisco==VISCO_Elasticity){ // This is dummy particle method. Soleimani method not yet implemented here for GPU
          tau_sum_xx_xy.x=taup1_xx_xy.x/pow(velrhop1.w,2) + taup2_xx_xy.x/pow(velrhop2.w,2);
          tau_sum_xx_xy.y=taup1_xx_xy.y/pow(velrhop1.w,2) + taup2_xx_xy.y/pow(velrhop2.w,2);
          tau_sum_xz_yy.x=taup1_xz_yy.x/pow(velrhop1.w,2) + taup2_xz_yy.x/pow(velrhop2.w,2);
          tau_sum_xz_yy.y=taup1_xz_yy.y/pow(velrhop1.w,2) + taup2_xz_yy.y/pow(velrhop2.w,2);
          tau_sum_yz_zz.x=taup1_yz_zz.x/pow(velrhop1.w,2) + taup2_yz_zz.x/pow(velrhop2.w,2);
          tau_sum_yz_zz.y=taup1_yz_zz.y/pow(velrhop1.w,2) + taup2_yz_zz.y/pow(velrhop2.w,2);

          float taux=(tau_sum_xx_xy.x*frx+tau_sum_xx_xy.y*fry+tau_sum_xz_yy.x*frz); // as per symetric tensor grad
          float tauy=(tau_sum_xx_xy.y*frx+tau_sum_xz_yy.y*fry+tau_sum_yz_zz.x*frz);
          float tauz=(tau_sum_xz_yy.x*frx+tau_sum_yz_zz.x*fry+tau_sum_yz_zz.y*frz); 
          //store acceleration
          acep1.x+=taux*massp2; acep1.y+=tauy*massp2; acep1.z+=tauz*massp2;
        }else{
          tau_sum_xx_xy.x=taup1_xx_xy.x+taup2_xx_xy.x; tau_sum_xx_xy.y=taup1_xx_xy.y+taup2_xx_xy.y;	tau_sum_xz_yy.x=taup1_xz_yy.x+taup2_xz_yy.x;
          tau_sum_xz_yy.y=taup1_xz_yy.y+taup2_xz_yy.y;	tau_sum_yz_zz.x=taup1_yz_zz.x+taup2_yz_zz.x;
          tau_sum_yz_zz.y=taup1_yz_zz.y+taup2_yz_zz.y;

          float taux=(tau_sum_xx_xy.x*frx+tau_sum_xx_xy.y*fry+tau_sum_xz_yy.x*frz)/(velrhop2.w);
          float tauy=(tau_sum_xx_xy.y*frx+tau_sum_xz_yy.y*fry+tau_sum_yz_zz.x*frz)/(velrhop2.w);
          float tauz=(tau_sum_xz_yy.x*frx+tau_sum_yz_zz.x*fry+tau_sum_yz_zz.y*frz)/(velrhop2.w);
          //store stresses
          massp2=(USE_FLOATING ? ftmassp2 : massp2);
          acep1.x+=taux*massp2; acep1.y+=tauy*massp2; acep1.z+=tauz*massp2;
        }
      }
    }
  }
}

__global__ void KerComputePress_NN(unsigned np,unsigned npb,float2 *tauff, float *pressg){
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x;
  if(p<(np-npb)){
    const unsigned p1=p+npb;
    // tau.xx, tau.yy, tau.zz
    pressg[p1] = -(tauff[p1*3].x + tauff[p1*3+1].y + tauff[p1*3+2].y)/3;
  }
}

//------------------------------------------------------------------------------
/// Fluid-Fluid interaction to diffuse stress oscillation based on Feng et al., 2021.
/// "Large deformation analysis of granular materials with stabilized and noise-free stress treatment in SPH"
//------------------------------------------------------------------------------
template<TpKernel tker,TpFtMode ftmode,TpVisco tvisco,bool symm>
__device__ void GetStressDiffusion(const unsigned &pini,const unsigned &pfin
  ,const float *ftomassp,float2 *tauff,const float4 *poscell,const float4 *velrhop,const typecode *code
  ,const typecode pp1,bool ftp1,const float4 &pscellp1,float2 &taup1_diff_xx_xy,float2 &taup1_diff_xz_yy,float2 &taup1_diff_yz_zz
  ,float2 &taup1_xx_xy_old,float2 &taup1_xz_yy_old,float2 &taup1_yz_zz_old)
{
  for(int p2=pini; p2<pfin; p2++) {
    const float4 pscellp2=poscell[p2];
    float drx=pscellp1.x-pscellp2.x+CTE.poscellsize*(CEL_GetX(__float_as_int(pscellp1.w))-CEL_GetX(__float_as_int(pscellp2.w)));
    float dry=pscellp1.y-pscellp2.y+CTE.poscellsize*(CEL_GetY(__float_as_int(pscellp1.w))-CEL_GetY(__float_as_int(pscellp2.w)));
    float drz=pscellp1.z-pscellp2.z+CTE.poscellsize*(CEL_GetZ(__float_as_int(pscellp1.w))-CEL_GetZ(__float_as_int(pscellp2.w)));
    if(symm)dry=pscellp1.y+pscellp2.y+CTE.poscellsize*CEL_GetY(__float_as_int(pscellp2.w)); //<vs_syymmetry>
    const float rr2=drx*drx+dry*dry+drz*drz;
    if(rr2<=CTE.kernelsize2 && rr2>=ALMOSTZERO) {
      //-Computes kernel.
      const float fac=cufsph::GetKernel_Fac<tker>(rr2);
      const float frx=fac*drx,fry=fac*dry,frz=fac*drz; //-Gradients.

      //-Obtains mass of particle p2 for NN and if any floating bodies exist.
      const typecode cod=code[p2];
      const typecode pp2=CODE_GetTypeValue(cod); //<vs_non-Newtonian>
      float massp2, cbar;
      const float zeta = 0.1; // Coefficient used to control the magnitude of diffusion and normally take values as 0.1 for most application. 
      if(tvisco==VISCO_Hypoplasticity){
         massp2 = PHASEHYPO[pp2].mass; cbar=PHASEHYPO[pp2].Cs0; 
      }else if(tvisco==VISCO_Elasticity){
         massp2 = PHASEELASTIC[pp2].mass; cbar=PHASEELASTIC[pp2].Cs0;
      }else{
         massp2 = PHASEARRAY[pp2].mass; cbar=PHASEARRAY[pp2].Cs0;
      }
    
      //-Obtiene masa de particula p2 en caso de existir floatings.
      bool ftp2=false;         //-Indicates if it is floating. | Indica si es floating.
      float ftmassp2;    //-Contains mass of floating body or massf if fluid. | Contiene masa de particula floating o massp2 si es bound o fluid.
      bool compute=true; //-Deactivated when DEM is used and is float-float or float-bound. | Se desactiva cuando se usa DEM y es float-float o float-bound.
      if(USE_FLOATING) {
        const typecode cod=code[p2];
        ftp2=CODE_IsFloating(cod);
        ftmassp2=(ftp2 ? ftomassp[CODE_GetTypeValue(cod)] : massp2);
        compute=!(USE_FTEXTERNAL && ftp1&&(ftp2)); //-Deactivated when DEM or Chrono is used and is float-float or float-bound. | Se desactiva cuando se usa DEM o Chrono y es float-float o float-bound.
      }
      const float rhopp2 = velrhop[p2].w;
      const float dot = drx*frx+dry*fry+drz*frz;
      const float dot_rr2=dot/(rr2+CTE.eta2)*massp2/rhopp2;
      const float k0 = 0.33333; // k0 for lateral earth pressure, kept a constant for simplicity for now. Can be k0=(1-sin(interal friction angle))
      // currently only valid for gravity on z direction. gz = -9.81.
      const float SD = 2*zeta*CTE.kernelh*cbar*dot_rr2;
      //Stress tensor at the end of previous increment
      float2 taup2_xx_xy_old=tauff[p2*3];
      float2 taup2_xz_yy_old=tauff[p2*3+1];
      float2 taup2_yz_zz_old=tauff[p2*3+2];
 
      taup1_diff_xx_xy.x += SD*(taup1_xx_xy_old.x - taup2_xx_xy_old.x + k0*rhopp2*9.81*drz); //let g be -9.81 on z for now.
      taup1_diff_xx_xy.y += SD*(taup1_xx_xy_old.y - taup2_xx_xy_old.y);
      taup1_diff_xz_yy.x += SD*(taup1_xz_yy_old.x - taup2_xz_yy_old.x);
      taup1_diff_xz_yy.y += SD*(taup1_xz_yy_old.y - taup2_xz_yy_old.y + k0*rhopp2*9.81*drz);
      taup1_diff_yz_zz.x += SD*(taup1_yz_zz_old.x - taup2_yz_zz_old.x);
      taup1_diff_yz_zz.y += SD*(taup1_yz_zz_old.y - taup2_yz_zz_old.y + k0*rhopp2*9.81*drz);
     }
    }
}

//------------------------------------------------------------------------------
/// Interaction between particles for non-Newtonian models using the SPH approach with Const. Eq. Fluid/Float-Fluid/Float or Fluid/Float-Bound.
/// Includes Const. Eq. viscosity and normal/DEM floating bodies que utilizan el enfoque de la SPH Const. Eq..
///
/// Realiza interaccion entre particulas. Fluid/Float-Fluid/Float or Fluid/Float-Bound
/// Incluye visco artificial/laminar y floatings normales/dem.
//------------------------------------------------------------------------------
template<TpKernel tker,TpFtMode ftmode,TpVisco tvisco,bool symm>
__global__ void KerInteractionForcesFluid_NN_SPH_ConsEq(unsigned n,unsigned pinit,float *visco_eta
  ,int scelldiv,int4 nc,int3 cellzero,const int2 *begincell,unsigned cellfluid,const unsigned *dcell
  ,const float *ftomassp,float2 *tauff,const float4 *poscell,const float4 *velrhop
  ,const typecode *code,float3 *ace,int Zhan_loop, bool regularize)
{
  const unsigned p=blockIdx.x*blockDim.x+threadIdx.x; //-Number of particle.
  if(p<n) {
    unsigned p1=p+pinit;      //-Number of particle.			
    float3 acep1=make_float3(0,0,0);
    // float weight_sum = 0;
    float visc=0;

    //-Obtains data of particle p1 in case there are floating bodies.
    //-Obtiene datos de particula p1 en caso de existir floatings.
    bool ftp1;       //-Indicates if it is floating. | Indica si es floating.
    const typecode cod=code[p1];
    if(USE_FLOATING) {
      const typecode cod=code[p1];
      ftp1=CODE_IsFloating(cod);
    }

    //-Obtains basic data of particle p1.
    const float4 pscellp1=poscell[p1];
    const float4 velrhop1=velrhop[p1];
    const bool rsymp1=(symm && CEL_GetPartY(__float_as_uint(pscellp1.w))==0); //<vs_syymmetry>
    //<vs_non-Newtonian>
    const typecode pp1=CODE_GetTypeValue(cod);
    // float visco_etap1=visco_eta[p1];

      //-Variables for tau.			
      float2 taup1_xx_xy=tauff[p1*3];
      float2 taup1_xz_yy=tauff[p1*3+1];
      float2 taup1_yz_zz=tauff[p1*3+2];

    //Stress Sum for regularization
    float4 taup1_xx_xy_xz_sum = make_float4(0,0,0,0);
    float3 taup1_yy_yz_zz = make_float3(0,0,0);

    float Hypo_miu, Elastic_miu, massp1;
    if(tvisco==VISCO_Hypoplasticity) Hypo_miu = PHASEHYPO[pp1].Hypo_wallfriction; massp1=PHASEHYPO[pp1].mass;
    if(tvisco==VISCO_Elasticity) Elastic_miu = PHASEELASTIC[pp1].Elastic_wallfriction; massp1=PHASEELASTIC[pp1].mass;
    if((tvisco!=VISCO_Hypoplasticity) && (tvisco!=VISCO_Elasticity)) massp1=PHASEARRAY[pp1].mass;
    //-Obtains neighborhood search limits.
    int ini1,fin1,ini2,fin2,ini3,fin3;
    cunsearch::InitCte(dcell[p1],scelldiv,nc,cellzero,ini1,fin1,ini2,fin2,ini3,fin3);

    //-Interaction with fluids.
    ini3+=cellfluid; fin3+=cellfluid;
    for(int c3=ini3; c3<fin3; c3+=nc.w)for(int c2=ini2; c2<fin2; c2+=nc.x) {
      unsigned pini,pfin=0; cunsearch::ParticleRange(c2,c3,ini1,fin1,begincell,pini,pfin);
      if(pfin) {
        KerInteractionForcesFluidBox_SPH_ConsEq<tker,ftmode,tvisco,false>(false,pini,pfin,ftomassp,tauff,poscell,velrhop,code,pp1,ftp1,pscellp1,velrhop1,taup1_xx_xy,taup1_xz_yy,taup1_yz_zz,acep1,visc,taup1_yy_yz_zz,p1,taup1_xx_xy_xz_sum,regularize);
        if(symm && rsymp1)	KerInteractionForcesFluidBox_SPH_ConsEq<tker,ftmode,tvisco,true>(false,pini,pfin,ftomassp,tauff,poscell,velrhop,code,pp1,ftp1,pscellp1,velrhop1,taup1_xx_xy,taup1_xz_yy,taup1_yz_zz,acep1,visc,taup1_yy_yz_zz,p1,taup1_xx_xy_xz_sum,regularize); //<vs_syymmetry>
      }
    }
    //-Interaction with boundaries.
    if (Zhan_loop == 0){
      ini3-=cellfluid; fin3-=cellfluid;
      for(int c3=ini3; c3<fin3; c3+=nc.w)for(int c2=ini2; c2<fin2; c2+=nc.x) {
        unsigned pini,pfin=0; cunsearch::ParticleRange(c2,c3,ini1,fin1,begincell,pini,pfin);
        if(pfin) {
          KerInteractionForcesFluidBox_SPH_ConsEq<tker,ftmode,tvisco,false>(true,pini,pfin,ftomassp,tauff,poscell,velrhop,code,pp1,ftp1,pscellp1,velrhop1,taup1_xx_xy,taup1_xz_yy,taup1_yz_zz,acep1,visc,taup1_yy_yz_zz,p1,taup1_xx_xy_xz_sum,regularize);
          if(symm && rsymp1)	KerInteractionForcesFluidBox_SPH_ConsEq<tker,ftmode,tvisco,true>(true,pini,pfin,ftomassp,tauff,poscell,velrhop,code,pp1,ftp1,pscellp1,velrhop1,taup1_xx_xy,taup1_xz_yy,taup1_yz_zz,acep1,visc,taup1_yy_yz_zz,p1,taup1_xx_xy_xz_sum,regularize); //<vs_syymmetry>
        }
      }
    }
    //-Stores results.
    if(acep1.x||acep1.y||acep1.z) {
      float3 r=ace[p1]; r.x+=acep1.x; r.y+=acep1.y; r.z+=acep1.z; ace[p1]=r;
      // auxnn[p1] = visco_etap1; // to be used if an auxilary is needed.
    }
    //if (regularize){ // regularize method 1
    //  tauff[p1*3]  =make_float2(taup1_xx_xy_xz_sum.x/taup1_xx_xy_xz_sum.w, taup1_xx_xy_xz_sum.y/taup1_xx_xy_xz_sum.w);
    //  tauff[p1*3+1]=make_float2(taup1_xx_xy_xz_sum.z/taup1_xx_xy_xz_sum.w, taup1_yy_yz_zz.x/taup1_xx_xy_xz_sum.w);
    //  tauff[p1*3+2]=make_float2(taup1_yy_yz_zz.y/taup1_xx_xy_xz_sum.w, taup1_yy_yz_zz.z/taup1_xx_xy_xz_sum.w);
    //}
  }
}

//==============================================================================
/// Perform interaction between particles for the SPH approcach using the Const. Eq.: 
/// Fluid/Float-Bound, particullarly for Zhan's method 
//==============================================================================
template<TpKernel tker,TpFtMode ftmode,TpVisco tvisco,TpSlipMode slipmode,bool symm>
__global__ void KerInteractionForcesFluid_NN_SPH_ConsEq_Zhan_bound(unsigned n,unsigned pinit,float *visco_eta
  ,int scelldiv,int4 nc,int3 cellzero,const int2 *begincell,const unsigned *dcell
  ,const float *ftomassp,float2 *tauff,const float4 *poscell,float4 *velrhop
  ,const typecode *code,float3 *ace, float3 *boundnormal,bool *boundCorner,float3 *Force,double dt)
{
  const unsigned p=blockIdx.x*blockDim.x+threadIdx.x; //-Number of particle.
  if(p<n) {
    unsigned p1=p+pinit;      //-Number of particle.			
    float3 acep1=make_float3(0,0,0);
    float3 Forcep2=make_float3(0,0,0);
    float weight_sum = 0;
    float visc=0;

    //-Obtains data of particle p1 in case there are floating bodies.
    //-Obtiene datos de particula p1 en caso de existir floatings.
    bool ftp1;       //-Indicates if it is floating. | Indica si es floating.
    const typecode cod=code[p1];
    if(USE_FLOATING) {
      const typecode cod=code[p1];
      ftp1=CODE_IsFloating(cod);
    }

    //-Obtains basic data of particle p1.
    const float4 pscellp1=poscell[p1];
    const float4 velrhop1=velrhop[p1];
    // const bool rsymp1=(symm && CEL_GetPartY(__float_as_uint(pscellp1.w))==0); //<vs_syymmetry>
    //<vs_non-Newtonian>
    const typecode pp1=CODE_GetTypeValue(cod);
    // float visco_etap1=visco_eta[p1];

    //-Variables for tau.			
    // float2 taup1_xx_xy=tauff[p1*3];
    // float2 taup1_xz_yy=tauff[p1*3+1];
    // float2 taup1_yz_zz=tauff[p1*3+2];

    float Hypo_miu, Elastic_miu, massp1;
    if(tvisco==VISCO_Hypoplasticity){
      Hypo_miu = PHASEHYPO[pp1].Hypo_wallfriction; 
      massp1=PHASEHYPO[pp1].mass;
      }
    if(tvisco==VISCO_Elasticity){
      Elastic_miu = PHASEELASTIC[pp1].Elastic_wallfriction; 
      massp1=PHASEELASTIC[pp1].mass;
    }
    if((tvisco!=VISCO_Hypoplasticity) && (tvisco!=VISCO_Elasticity)) massp1=PHASEARRAY[pp1].mass;
    
    //-Obtains neighborhood search limits.
    int ini1,fin1,ini2,fin2,ini3,fin3;
    cunsearch::InitCte(dcell[p1],scelldiv,nc,cellzero,ini1,fin1,ini2,fin2,ini3,fin3);
    float rr2; float drx; float dry; float drz; int p2;
    int p2_nearest = 0; // p2_nearest: nearest boundary particle to p1
    drx=float(pscellp1.x - poscell[p2_nearest].x+CTE.poscellsize*(CEL_GetX(__float_as_int(pscellp1.w))-CEL_GetX(__float_as_int(poscell[p2_nearest].w))));
    dry=float(pscellp1.y - poscell[p2_nearest].y+CTE.poscellsize*(CEL_GetY(__float_as_int(pscellp1.w))-CEL_GetY(__float_as_int(poscell[p2_nearest].w))));
    drz=float(pscellp1.z - poscell[p2_nearest].z+CTE.poscellsize*(CEL_GetZ(__float_as_int(pscellp1.w))-CEL_GetZ(__float_as_int(poscell[p2_nearest].w))));
    rr2=drx*drx+dry*dry+drz*drz;
    // float rr2_temp=rr2;

    //-Interaction with boundaries, Zhan's method.
    for(int c3=ini3; c3<fin3; c3+=nc.w)for(int c2=ini2; c2<fin2; c2+=nc.x) {
      unsigned pini,pfin=0; cunsearch::ParticleRange(c2,c3,ini1,fin1,begincell,pini,pfin);
      // Find nearest boundary particle p2 with regard to material particle p1
      for(int p2=pini; p2<pfin; p2++) {
        drx=float(pscellp1.x - poscell[p2].x+CTE.poscellsize*(CEL_GetX(__float_as_int(pscellp1.w))-CEL_GetX(__float_as_int(poscell[p2].w))));
        dry=float(pscellp1.y - poscell[p2].y+CTE.poscellsize*(CEL_GetY(__float_as_int(pscellp1.w))-CEL_GetY(__float_as_int(poscell[p2].w))));
        float drz=float(pscellp1.z - poscell[p2].z+CTE.poscellsize*(CEL_GetZ(__float_as_int(pscellp1.w))-CEL_GetZ(__float_as_int(poscell[p2].w))));
        float rr2_temp = drx*drx+dry*dry+drz*drz;
        if(rr2_temp<rr2){ 
          p2_nearest = p2; 
          rr2 = rr2_temp;
        }
      }
    }
    p2 = p2_nearest;
    drx=float(pscellp1.x - poscell[p2].x + CTE.poscellsize*(CEL_GetX(__float_as_int(pscellp1.w))-CEL_GetX(__float_as_int(poscell[p2].w))));
    dry=float(pscellp1.y - poscell[p2].y + CTE.poscellsize*(CEL_GetY(__float_as_int(pscellp1.w))-CEL_GetY(__float_as_int(poscell[p2].w))));
    drz=float(pscellp1.z - poscell[p2].z + CTE.poscellsize*(CEL_GetZ(__float_as_int(pscellp1.w))-CEL_GetZ(__float_as_int(poscell[p2].w)))); 
    
    if(rr2<=CTE.kernelsize2 && rr2>=ALMOSTZERO) {
      //-Computes kernel.
      const float fac=cufsph::GetKernel_Fac<tker>(rr2);
      const float frx=fac*drx,fry=fac*dry,frz=fac*drz; //-Gradients.

      //-Obtains mass of particle p2 for NN and if any floating bodies exist.
      const typecode cod=code[p2];
      float massp2 = CTE.massb;

      //-Obtiene masa de particula p2 en caso de existir floatings.
      // bool ftp2=false;         //-Indicates if it is floating. | Indica si es floating.
      // float ftmassp2;    //-Contains mass of floating body or massf if fluid. | Contiene masa de particula floating o massp2 si es bound o fluid.
      bool compute=true; //-Deactivated when DEM is used and is float-float or float-bound. | Se desactiva cuando se usa DEM y es float-float o float-bound.
      if(USE_FLOATING) {
        //const typecode cod=code[p2];
        //ftp2=CODE_IsFloating(cod);
        //ftmassp2=(ftp2 ? ftomassp[CODE_GetTypeValue(cod)] : massp2);
        //compute=!(USE_FTEXTERNAL && ftp1&&(boundp2||ftp2)); //-Deactivated when DEM or Chrono is used and is float-float or float-bound. | Se desactiva cuando se usa DEM o Chrono y es float-float o float-bound.
      }

      float4 velrhop2=velrhop[p2];
      if(symm)velrhop2.y=-velrhop2.y; //<vs_syymmetry>

      //-velocity dvx.
      const float dvx=velrhop1.x-velrhop2.x,dvy=velrhop1.y-velrhop2.y,dvz=velrhop1.z-velrhop2.z;
      //===== Viscosity ===== 
      if(compute) {
        const float dot=drx*dvx+dry*dvy+drz*dvz;
        const float dot_rr2=dot/(rr2+CTE.eta2);
        visc=max(dot_rr2,visc);  //ViscDt=max(dot/(rr2+Eta2),ViscDt);

        //<vs_non-Newtonian>				
        float2 tau_sum_xx_xy=make_float2(0,0);
        float2 tau_sum_xz_yy=make_float2(0,0);
        float2 tau_sum_yz_zz=make_float2(0,0);
        // float2 taup2_xx_xy=tauff[p2*3];
        // float2 taup2_xz_yy=tauff[p2*3+1];
        // float2 taup2_yz_zz=tauff[p2*3+2];

        if ((tvisco == VISCO_Hypoplasticity) || (tvisco == VISCO_Elasticity)){
          float3 normal=make_float3(0,0,0);
          double ds;
          float miu = (tvisco == VISCO_Hypoplasticity ? Hypo_miu : Elastic_miu);
          if(boundCorner[p2]){ // Redefine normal for corner bound particles
            ds = sqrt(rr2);
            if(ds!=0) normal.x = drx/ds; normal.y = dry/ds; normal.z = drz/ds;
          }
          else{
            normal = boundnormal[p2];  
            const float normal_mag = sqrt(normal.x*normal.x+normal.y*normal.y+normal.z*normal.z);
            if (normal_mag!=0){
            normal.x = normal.x/normal_mag;
            normal.y = normal.y/normal_mag;
            normal.z = normal.z/normal_mag;
            }
            ds = abs(drx*normal.x + dry*normal.y + drz*normal.z);
          }
          if((ds < CTE.dp) & ((dvx*normal.x + dvy*normal.y + dvz*normal.z ) < 0)){ 
            // Normal contact force magnitude from boundary to soil particle
            //float t_n_mag = abs((normal.x*dvx + normal.y*dvy + normal.z*dvz)/(dt/massp2 + dt/massp1)); //This is when p2 is floating boundary
            float t_n_mag = abs((normal.x*dvx + normal.y*dvy + normal.z*dvz)/(dt/massp1));
            float3 t_n = make_float3(0,0,0); 
            t_n.x = t_n_mag*normal.x; t_n.y = t_n_mag*normal.y; t_n.z = t_n_mag*normal.z;
            // Shear contact force magnitude from boundary to soil particles
            float3 t_s = make_float3(0,0,0);
            //t_s.x = -dvx/(dt/massp2 + dt/massp1) - t_n.x; This is when p2 is floating boundary
            //t_s.y = -dvy/(dt/massp2 + dt/massp1) - t_n.y;
            //t_s.z = -dvz/(dt/massp2 + dt/massp1) - t_n.z;
            if(slipmode == SLIP_FreeSlip_Zhan){
              t_s.x = 0; t_s.y=0; t_s.z=0;
            }else{ // frictional slip or no-slip
            t_s.x = -dvx/(dt/massp1) - t_n.x;
            t_s.y = -dvy/(dt/massp1) - t_n.y;
            t_s.z = -dvz/(dt/massp1) - t_n.z;
            }
            float t_s_mag = sqrt(t_s.x*t_s.x + t_s.y*t_s.y + t_s.z*t_s.z);
            if((t_s_mag >= miu*t_n_mag) && (t_s_mag!=0) && slipmode == SLIP_Friction_Zhan){
              t_s.x = miu*t_n_mag/t_s_mag*t_s.x;
              t_s.y = miu*t_n_mag/t_s_mag*t_s.y;
              t_s.z = miu*t_n_mag/t_s_mag*t_s.z;
              t_s_mag = sqrt(t_s.x*t_s.x + t_s.y*t_s.y + t_s.z*t_s.z);
            }
            //store acceleration
            acep1.x= t_n.x + t_s.x; // Here acep1 is actullay in [N], not [N/kg]
            acep1.y= t_n.y + t_s.y;
            acep1.z= t_n.z + t_s.z;
            Forcep2.x = -acep1.x/massp1*massp2; //Here force is in [N/kg*kg] = [N]
            Forcep2.y = -acep1.y/massp1*massp2; // This force is in between a single p1-p2 pair
            Forcep2.z = -acep1.z/massp1*massp2;
          }   
        }
      }
    }
    //-Sum results together. | Almacena resultados.
    if(acep1.x||acep1.y||acep1.z) {
      if ((tvisco==VISCO_Hypoplasticity || tvisco==VISCO_Elasticity)){
        // Update velocity to n+1/2 step, not sure wether to include weight_sum:  acep1.x=acep1.x/massp1/weight_sum??
        acep1.x = acep1.x/massp1;
        acep1.y = acep1.y/massp1;
        acep1.z = acep1.z/massp1;
        velrhop[p1].x += dt*acep1.x; //From predicted to corrected velocity at n+1/2 step.
        velrhop[p1].y += dt*acep1.y;
        velrhop[p1].z += dt*acep1.z;
      }
      ace[p1].x=ace[p1].x+acep1.x;  ace[p1].y=ace[p1].y+acep1.y;  ace[p1].z=ace[p1].z+acep1.z;
      Force[p2].x=Force[p2].x+Forcep2.x;  Force[p2].y=Force[p2].y+Forcep2.y;  Force[p2].z=Force[p2].z+Forcep2.z;
    }
    //if (regularize){
    //  tauff[p1*3]  =make_float2(taup1_xx_xy_xz_sum.x/taup1_xx_xy_xz_sum.w, taup1_xx_xy_xz_sum.y/taup1_xx_xy_xz_sum.w);
    //  tauff[p1*3+1]=make_float2(taup1_xx_xy_xz_sum.z/taup1_xx_xy_xz_sum.w, taup1_yy_yz_zz.x/taup1_xx_xy_xz_sum.w);
    //  tauff[p1*3+2]=make_float2(taup1_yy_yz_zz.y/taup1_xx_xy_xz_sum.w, taup1_yy_yz_zz.z/taup1_xx_xy_xz_sum.w);
    //}
    //if (p1 == 113){
    // printf("%s \n","In conseq_bound_zhan");
    //  printf("acep1: %f %f %f\n",acep1.x, acep1.y, acep1.z);
    //}
  }
}
//==============================================================================
/// Calculates the strain rate tensor and effective viscocity for each particle for non-Newtonian models.
/// Calcula el tensor de la velocidad de deformacion y la viscosidad efectiva para cada particula para modelos no-Newtonianos.
//==============================================================================
template<TpKernel tker,TpFtMode ftmode,TpVisco tvisco,bool symm>
__global__ void KerInteractionForcesFluid_NN_SPH_Visco_Stress_tensor(unsigned n,unsigned pinit,float *visco_eta
  ,int scelldiv,int4 nc,int3 cellzero,const int2 *begincell,unsigned cellfluid,const unsigned *dcell,
  const float *ftomassp,float2 *tauff,const float4 *poscell,float4 *velrhop,float *void_ratio,const typecode *code, float3 *gradvelff
  ,double dt, bool stop, bool regularize)
{
  const unsigned p=blockIdx.x*blockDim.x+threadIdx.x; //-Number of particle.
  if(p<n) {
    unsigned p1=p+pinit;      //-Number of particle.
        //-Obtains basic data of particle p1.
    //-Obtains data of particle p1 in case there are floating bodies.
    //-Obtiene datos de particula p1 en caso de existir floatings.
    bool ftp1;       //-Indicates if it is floating. | Indica si es floating.
    const typecode cod=code[p1];
    if(USE_FLOATING) {
      const typecode cod=code[p1];
      ftp1=CODE_IsFloating(cod);
    }

    const float4 pscellp1=poscell[p1];
    const float4 velrhop1=velrhop[p1];
    float rhopp1 = velrhop[p1].w;
    const bool rsymp1=(symm && CEL_GetPartY(__float_as_uint(pscellp1.w))==0); //<vs_syymmetry>
    //<vs_non-Newtonian>
    const typecode pp1=CODE_GetTypeValue(cod);

    //update stress tensor
    float2 taup1_xx_xy=make_float2(0,0);
    float2 taup1_xz_yy=make_float2(0,0);
    float2 taup1_yz_zz=make_float2(0,0);
    //for stress diffusion
    float2 taup1_diff_xx_xy=make_float2(0,0);
    float2 taup1_diff_xz_yy=make_float2(0,0);
    float2 taup1_diff_yz_zz=make_float2(0,0);
    //Stress tensor at the end of previous increment
    float2 taup1_xx_xy_old=tauff[p1*3];
    float2 taup1_xz_yy_old=tauff[p1*3+1];
    float2 taup1_yz_zz_old=tauff[p1*3+2];

    //Strain rate tensor 
    float2 dtsrp1_xx_xy=make_float2(0,0);
    float2 dtsrp1_xz_yy=make_float2(0,0);
    float2 dtsrp1_yz_zz=make_float2(0,0);
    //spin rate tensor
    float3 dtspinratep1 =  make_float3(0,0,0);
    //********************************************************
    // Stress diffusion
    if(regularize){
      //-Obtains neighborhood search limits.
      int ini1,fin1,ini2,fin2,ini3,fin3;
      cunsearch::InitCte(dcell[p1],scelldiv,nc,cellzero,ini1,fin1,ini2,fin2,ini3,fin3);
      
      //-Interaction with fluids.
      ini3+=cellfluid; fin3+=cellfluid;
      for(int c3=ini3; c3<fin3; c3+=nc.w)for(int c2=ini2; c2<fin2; c2+=nc.x) {
        unsigned pini,pfin=0; cunsearch::ParticleRange(c2,c3,ini1,fin1,begincell,pini,pfin);
        if(pfin) {
          GetStressDiffusion<tker,ftmode,tvisco,false>(pini,pfin,ftomassp,tauff,poscell,velrhop,code,pp1,ftp1,pscellp1,taup1_diff_xx_xy,taup1_diff_xz_yy,taup1_diff_yz_zz,taup1_xx_xy_old,taup1_xz_yy_old,taup1_yz_zz_old);
          if(symm && rsymp1)	GetStressDiffusion<tker,ftmode,tvisco,true>(pini,pfin,ftomassp,tauff,poscell,velrhop,code,pp1,ftp1,pscellp1,taup1_diff_xx_xy,taup1_diff_xz_yy,taup1_diff_yz_zz,taup1_xx_xy_old,taup1_xz_yy_old,taup1_yz_zz_old); //<vs_syymmetry>
        }
      }
    }
    //********************************************************  
    if ((tvisco == VISCO_Hypoplasticity) || (tvisco == VISCO_Elasticity)){
      //-Variables for gradients.
      float3 grap1_xx_xy_xz,grap1_yx_yy_yz,grap1_zx_zy_zz;
      grap1_xx_xy_xz=gradvelff[p1*3];
      grap1_yx_yy_yz=gradvelff[p1*3+1];
      grap1_zx_zy_zz=gradvelff[p1*3+2];

      GetStrainSpinRateTensor(grap1_xx_xy_xz,grap1_yx_yy_yz,grap1_zx_zy_zz,dtsrp1_xx_xy,dtsrp1_xz_yy,dtsrp1_yz_zz,dtspinratep1);
      /*
      if(isnan(dtsrp1_xx_xy.x) || isinf(dtsrp1_xx_xy.x)) dtsrp1_xx_xy.x = 0;
      if(isnan(dtsrp1_xx_xy.y) || isinf(dtsrp1_xx_xy.y)) dtsrp1_xx_xy.y = 0;
      if(isnan(dtsrp1_xz_yy.x) || isinf(dtsrp1_xz_yy.x)) dtsrp1_xz_yy.x = 0;
      if(isnan(dtsrp1_xz_yy.y) || isinf(dtsrp1_xz_yy.y)) dtsrp1_xz_yy.y = 0;
      if(isnan(dtsrp1_yz_zz.x) || isinf(dtsrp1_yz_zz.x)) dtsrp1_yz_zz.x = 0;
      if(isnan(dtsrp1_yz_zz.y) || isinf(dtsrp1_yz_zz.y)) dtsrp1_yz_zz.y = 0;
      */
      if (tvisco == VISCO_Hypoplasticity){
      //void ratio at the end of previous increment
        float voidRatio = void_ratio[p1];
        const typecode pp1=CODE_GetTypeValue(code[p1]); //phase information
        const float Hypo_angle = PHASEHYPO[pp1].Hypo_angle; ///<  Internal friction angle
        const float Hypo_hs= PHASEHYPO[pp1].Hypo_hs;    ///< hs
        const float Hypo_n= PHASEHYPO[pp1].Hypo_n;    ///< n
        const float Hypo_ed0= PHASEHYPO[pp1].Hypo_ed0;    ///< ed_0
        const float Hypo_ec0= PHASEHYPO[pp1].Hypo_ec0;    ///< ec_0
        const float Hypo_ei0= PHASEHYPO[pp1].Hypo_ei0;    ///< ei_0 
        const float Hypo_alpha= PHASEHYPO[pp1].Hypo_alpha;    ///< alpha
        const float Hypo_beta= PHASEHYPO[pp1].Hypo_beta;    ///< beta

        GetStressTensorHypo(dtsrp1_xx_xy, dtsrp1_xz_yy, dtsrp1_yz_zz, dtspinratep1,
        taup1_xx_xy_old, taup1_xz_yy_old, taup1_yz_zz_old,
        taup1_xx_xy, taup1_xz_yy, taup1_yz_zz,
        taup1_diff_xx_xy, taup1_diff_xz_yy, taup1_diff_yz_zz, 
        voidRatio, dt, Hypo_angle, Hypo_hs, Hypo_n,
        Hypo_ed0, Hypo_ec0, Hypo_ei0, Hypo_alpha, Hypo_beta,regularize,stop);
        //-Stores results.
        void_ratio[p1] = voidRatio;
        rhopp1 = PHASEHYPO[pp1].Hypo_rhoparticle/(1 + voidRatio);
        velrhop[p1].w = rhopp1;
        //if (isnan(voidRatio) || isnan(taup1_xx_xy.x) || isnan(taup1_xx_xy.y) || isnan(taup1_xz_yy.x) || isnan(taup1_xz_yy.y) || isnan(taup1_yz_zz.x) || isnan(taup1_yz_zz.y)){
        //  stop=true;
       //}
      }else if (tvisco == VISCO_Elasticity){
        const typecode pp1=CODE_GetTypeValue(code[p1]); //phase information
        const float lameparm1 = PHASEELASTIC[pp1].lameparm1;
        const float lameparm2 = PHASEELASTIC[pp1].lameparm2;
        GetStressTensorElastic(dtsrp1_xx_xy, dtsrp1_xz_yy, dtsrp1_yz_zz, dtspinratep1, taup1_xx_xy_old,
        taup1_xz_yy_old, taup1_yz_zz_old, taup1_xx_xy, taup1_xz_yy, taup1_yz_zz,
        taup1_diff_xx_xy, taup1_diff_xz_yy, taup1_diff_yz_zz, dt, lameparm2, lameparm1,regularize);
      }
        tauff[p1*3]=make_float2(taup1_xx_xy.x,taup1_xx_xy.y);
        tauff[p1*3+1]=make_float2(taup1_xz_yy.x,taup1_xz_yy.y);
        tauff[p1*3+2]=make_float2(taup1_yz_zz.x,taup1_yz_zz.y);
    }else{
      //<vs_non-Newtonian>
      //float visco_etap1=visco_eta[p1];;
      //Strain rate tensor 
      //float2 dtsrp1_xx_xy=d_tensorff[p1*3];
      //float2 dtsrp1_xz_yy=d_tensorff[p1*3+1];
      //float2 dtsrp1_yz_zz=d_tensorff[p1*3+2];

      //float I_t,II_t; float J1_t,J2_t; float tau_tensor_magn;
      //GetStressTensor_sym(dtsrp1_xx_xy,dtsrp1_xz_yy,dtsrp1_yz_zz,visco_etap1,I_t,II_t,J1_t,J2_t,tau_tensor_magn,taup1_xx_xy,taup1_xz_yy,taup1_yz_zz);
      //-Stores results.
      //float2 rg;
      //rg=tauff[p1*3];    rg=make_float2(rg.x+taup1_xx_xy.x,rg.y+taup1_xx_xy.y);  tauff[p1*3]=rg;
      //rg=tauff[p1*3+1];  rg=make_float2(rg.x+taup1_xz_yy.x,rg.y+taup1_xz_yy.y);  tauff[p1*3+1]=rg;
      //rg=tauff[p1*3+2];  rg=make_float2(rg.x+taup1_yz_zz.x,rg.y+taup1_yz_zz.y);  tauff[p1*3+2]=rg;
    }
  }
}

//------------------------------------------------------------------------------
/// Interaction of a particle with a set of particles for non-Newtonian models using the SPH approach. (Fluid/Float-Fluid/Float/Bound)
/// Realiza la interaccion de una particula con un conjunto de ellas para modelos no-Newtonianos que utilizan el enfoque de la SPH. (Fluid/Float-Fluid/Float/Bound)
//------------------------------------------------------------------------------
template<TpKernel tker,TpFtMode ftmode,TpVisco tvisco,bool symm>
__device__ void KerInteractionForcesFluidBox_SPH_Morris(bool boundp2,unsigned p1
  ,const unsigned &pini,const unsigned &pfin,float visco,float *visco_eta
  ,const float *ftomassp
  ,const float4 *poscell,const float4 *velrhop
  ,const typecode *code,const unsigned *idp
  ,const typecode pp1,bool ftp1
  ,const float4 &pscellp1,const float4 &velrhop1
  ,float3 &acep1,float &visc,float &visco_etap1)
{
  for(int p2=pini; p2<pfin; p2++) {
    const float4 pscellp2=poscell[p2];
    float drx=pscellp1.x-pscellp2.x+CTE.poscellsize*(CEL_GetX(__float_as_int(pscellp1.w))-CEL_GetX(__float_as_int(pscellp2.w)));
    float dry=pscellp1.y-pscellp2.y+CTE.poscellsize*(CEL_GetY(__float_as_int(pscellp1.w))-CEL_GetY(__float_as_int(pscellp2.w)));
    float drz=pscellp1.z-pscellp2.z+CTE.poscellsize*(CEL_GetZ(__float_as_int(pscellp1.w))-CEL_GetZ(__float_as_int(pscellp2.w)));
    if(symm)dry=pscellp1.y+pscellp2.y+CTE.poscellsize*CEL_GetY(__float_as_int(pscellp2.w)); //<vs_syymmetry>
    const float rr2=drx*drx+dry*dry+drz*drz;
    if(rr2<=CTE.kernelsize2 && rr2>=ALMOSTZERO) {
      //-Computes kernel.
      const float fac=cufsph::GetKernel_Fac<tker>(rr2);
      const float frx=fac*drx,fry=fac*dry,frz=fac*drz; //-Gradients.

      //-Obtains mass of particle p2 for NN and if any floating bodies exist.
      const typecode cod=code[p2];
      const typecode pp2=(boundp2 ? pp1 : CODE_GetTypeValue(cod)); //<vs_non-Newtonian>
      float massp2=(boundp2 ? CTE.massb : PHASEARRAY[pp2].mass); //massp2 not neccesary to go in _Box function
      //Note if you masses are very different more than a ratio of 1.3 then: massp2 = (boundp2 ? PHASEARRAY[pp1].mass : PHASEARRAY[pp2].mass);

      bool ftp2=false;        //-Indicates if it is floating. | Indica si es floating.
      float ftmassp2;						//-Contains mass of floating body or massf if fluid. | Contiene masa de particula floating o massp2 si es bound o fluid.
      bool compute=true;			//-Deactivated when DEM is used and is float-float or float-bound. | Se desactiva cuando se usa DEM y es float-float o float-bound.
      if(USE_FLOATING) {
        const typecode cod=code[p2];
        ftp2=CODE_IsFloating(cod);
        ftmassp2=(ftp2 ? ftomassp[CODE_GetTypeValue(cod)] : massp2);
        compute=!(USE_FTEXTERNAL && ftp1&&(boundp2||ftp2)); //-Deactivated when DEM or Chrono is used and is float-float or float-bound. | Se desactiva cuando se usa DEM o Chrono y es float-float o float-bound.
      }

      float4 velrhop2=velrhop[p2];
      if(symm)velrhop2.y=-velrhop2.y; //<vs_syymmetry>

      //-velocity dvx.
      float dvx=velrhop1.x-velrhop2.x,dvy=velrhop1.y-velrhop2.y,dvz=velrhop1.z-velrhop2.z;
      if(boundp2) { //this applies no slip on stress tensor
        dvx=2.f*velrhop1.x; dvy=2.f*velrhop1.y; dvz=2.f*velrhop1.z;  //fomraly I should use the moving BC vel as ug=2ub-uf
      }
      const float cbar=max(PHASEARRAY[pp2].Cs0,PHASEARRAY[pp2].Cs0); //get max Cs0 of phases

      //===== Viscosity ===== 
      if(compute) {
        const float dot=drx*dvx+dry*dvy+drz*dvz;
        const float dot_rr2=dot/(rr2+CTE.eta2);
        visc=max(dot_rr2,visc);  //ViscDt=max(dot/(rr2+Eta2),ViscDt);
        //<vs_non-Newtonian>
        const float visco_NN=PHASECTE[pp2].visco;
        if(tvisco==VISCO_Artificial) {//-Artificial viscosity.
          if(dot<0) {
            const float amubar=CTE.kernelh*dot_rr2;  //amubar=CTE.kernelh*dot/(rr2+CTE.eta2);
            const float robar=(velrhop1.w+velrhop2.w)*0.5f;
            const float pi_visc=(-visco_NN*cbar*amubar/robar)*(USE_FLOATING ? ftmassp2 : massp2);
            acep1.x-=pi_visc*frx; acep1.y-=pi_visc*fry; acep1.z-=pi_visc*frz;
          }
        }
        else if(tvisco!=VISCO_Artificial) {//-Laminar viscosity.
          {//-Laminar contribution.
            //vel gradients
            float visco_etap2=visco_eta[p2];
            //Morris Operator
            if(boundp2)visco_etap2=visco_etap1;
            //Morris Operator
            const float temp=(visco_etap1+visco_etap2)/((rr2+CTE.eta2)*velrhop2.w);
            const float vtemp=(USE_FLOATING ? ftmassp2 : massp2)*temp*(drx*frx+dry*fry+drz*frz);
            acep1.x+=vtemp*dvx; acep1.y+=vtemp*dvy; acep1.z+=vtemp*dvz;
          }
          //-SPS turbulence model.
          //-SPS turbulence model is disabled in v5.0 NN version
        }
      }
    }
  }
}

//------------------------------------------------------------------------------
/// Interaction between particles for non-Newtonian models using the SPH approach. Fluid/Float-Fluid/Float or Fluid/Float-Bound.
/// Includes artificial/laminar viscosity and normal/DEM floating bodies.
///
/// Realiza interaccion entre particulas para modelos no-Newtonianos que utilizan el enfoque de la SPH. Fluid/Float-Fluid/Float or Fluid/Float-Bound
/// Incluye visco artificial/laminar y floatings normales/dem.
//------------------------------------------------------------------------------
template<TpKernel tker,TpFtMode ftmode,TpVisco tvisco,bool symm>
__global__ void KerInteractionForcesFluid_NN_SPH_Morris(unsigned n,unsigned pinit,float viscob,float viscof,float *visco_eta
  ,int scelldiv,int4 nc,int3 cellzero,const int2 *begincell,unsigned cellfluid,const unsigned *dcell
  ,const float *ftomassp,float *auxnn,const float4 *poscell,const float4 *velrhop
  ,const typecode *code,const unsigned *idp
  ,float3 *ace)
{
  const unsigned p=blockIdx.x*blockDim.x+threadIdx.x; //-Number of particle.
  if(p<n) {
    unsigned p1=p+pinit;      //-Number of particle.			
    float3 acep1=make_float3(0,0,0);
    float visc=0;

    //-Obtains data of particle p1 in case there are floating bodies.
    //-Obtiene datos de particula p1 en caso de existir floatings.
    bool ftp1;       //-Indicates if it is floating. | Indica si es floating.		
    const typecode cod=code[p1];
    if(USE_FLOATING) {
      const typecode cod=code[p1];
      ftp1=CODE_IsFloating(cod);
    }

    //-Obtains basic data of particle p1.
    const float4 pscellp1=poscell[p1];
    const float4 velrhop1=velrhop[p1];
    const bool rsymp1=(symm && CEL_GetPartY(__float_as_uint(pscellp1.w))==0); //<vs_syymmetry>

    //<vs_non-Newtonian>
    const typecode pp1=CODE_GetTypeValue(cod);
    float visco_etap1=visco_eta[p1];

    //-Obtains neighborhood search limits.
    int ini1,fin1,ini2,fin2,ini3,fin3;
    cunsearch::InitCte(dcell[p1],scelldiv,nc,cellzero,ini1,fin1,ini2,fin2,ini3,fin3);

    //-Interaction with fluids.
    ini3+=cellfluid; fin3+=cellfluid;
    for(int c3=ini3; c3<fin3; c3+=nc.w)for(int c2=ini2; c2<fin2; c2+=nc.x) {
      unsigned pini,pfin=0; cunsearch::ParticleRange(c2,c3,ini1,fin1,begincell,pini,pfin);
      if(pfin) {
        KerInteractionForcesFluidBox_SPH_Morris<tker,ftmode,tvisco,false>(false,p1,pini,pfin,viscof,visco_eta,ftomassp,poscell,velrhop,code,idp,pp1,ftp1,pscellp1,velrhop1,acep1,visc,visco_etap1);
        if(symm && rsymp1)	KerInteractionForcesFluidBox_SPH_Morris<tker,ftmode,tvisco,true>(false,p1,pini,pfin,viscof,visco_eta,ftomassp,poscell,velrhop,code,idp,pp1,ftp1,pscellp1,velrhop1,acep1,visc,visco_etap1);
      }
    }
    //-Interaction with boundaries.
    ini3-=cellfluid; fin3-=cellfluid;
    for(int c3=ini3; c3<fin3; c3+=nc.w)for(int c2=ini2; c2<fin2; c2+=nc.x) {
      unsigned pini,pfin=0; cunsearch::ParticleRange(c2,c3,ini1,fin1,begincell,pini,pfin);
      if(pfin) {
        KerInteractionForcesFluidBox_SPH_Morris<tker,ftmode,tvisco,false>(true,p1,pini,pfin,viscob,visco_eta,ftomassp,poscell,velrhop,code,idp,pp1,ftp1,pscellp1,velrhop1,acep1,visc,visco_etap1);
        if(symm && rsymp1)	KerInteractionForcesFluidBox_SPH_Morris<tker,ftmode,tvisco,true>(true,p1,pini,pfin,viscob,visco_eta,ftomassp,poscell,velrhop,code,idp,pp1,ftp1,pscellp1,velrhop1,acep1,visc,visco_etap1);
      }
    }
    //-Stores results.
    if(acep1.x||acep1.y||acep1.z) {
      float3 r=ace[p1]; r.x+=acep1.x; r.y+=acep1.y; r.z+=acep1.z; ace[p1]=r;
      //auxnn[p1] = visco_etap1; // to be used if an auxilary is needed.
    }
  }
}

//==============================================================================
/// Calculates the strain rate tensor and effective viscocity for each particle
/// Calcula el tensor de la velocidad de deformacion y la viscosidad efectiva para cada particula.
//==============================================================================
template<TpFtMode ftmode,TpVisco tvisco,bool symm>
__global__ void KerInteractionForcesFluid_NN_SPH_Visco_eta(unsigned n,unsigned pinit,float viscob,float *visco_eta,const float4 *velrhop
  ,int scelldiv,int4 nc,int3 cellzero,const int2 *begincell,unsigned cellfluid,const unsigned *dcell
  ,float2 *d_tensorff,float3 *w_tensorff, float3 *gradvelff
  ,const typecode *code,const unsigned *idp
  ,float *viscetadt)
{
  const unsigned p=blockIdx.x*blockDim.x+threadIdx.x; //-Number of particle.
  if(p<n) {
    unsigned p1=p+pinit;      //-Number of particle.
    //-Obtains basic data of particle p1.
    //const float4 pscellp1 = poscell[p1];
    //const float4 velrhop1 = velrhop[p1];
    //<vs_non-Newtonian>
    const typecode cod=code[p1];
    const typecode pp1=CODE_GetTypeValue(cod);
    float visco_etap1=0;

    //-Variables for gradients.
    float3 grap1_xx_xy_xz,grap1_yx_yy_yz,grap1_zx_zy_zz;
    grap1_xx_xy_xz=gradvelff[p1*3];
    grap1_yx_yy_yz=gradvelff[p1*3+1];
    grap1_zx_zy_zz=gradvelff[p1*3+2];
    if (tvisco == VISCO_Hypoplasticity){
      //Strain rate tensor 
      float2 dtsrp1_xx_xy=make_float2(0,0);
      float2 dtsrp1_xz_yy=make_float2(0,0);
      float2 dtsrp1_yz_zz=make_float2(0,0);
      float3 dtspinratep1 =  make_float3(0,0,0);
      GetStrainSpinRateTensor(grap1_xx_xy_xz,grap1_yx_yy_yz,grap1_zx_zy_zz,dtsrp1_xx_xy,dtsrp1_xz_yy,dtsrp1_yz_zz,dtspinratep1);
      d_tensorff[p1*3] =make_float2(dtsrp1_xx_xy.x, dtsrp1_xx_xy.y);
      d_tensorff[p1*3+1]=make_float2(dtsrp1_xz_yy.x, dtsrp1_xz_yy.y);
      d_tensorff[p1*3+2]=make_float2(dtsrp1_yz_zz.x, dtsrp1_yz_zz.y);
      w_tensorff[p1]=make_float3(dtspinratep1.x, dtspinratep1.y, dtspinratep1.z);
    }else{
      //Strain rate tensor 
      float2 dtsrp1_xx_xy=make_float2(0,0);
      float2 dtsrp1_xz_yy=make_float2(0,0);
      float2 dtsrp1_yz_zz=make_float2(0,0);
      float div_D_tensor=0; float D_tensor_magn=0;
      float I_D,II_D; float J1_D,J2_D;
      GetStrainRateTensor_tsym(grap1_xx_xy_xz,grap1_yx_yy_yz,grap1_zx_zy_zz,I_D,II_D,J1_D,J2_D,div_D_tensor,D_tensor_magn,dtsrp1_xx_xy,dtsrp1_xz_yy,dtsrp1_yz_zz);

      //Effective viscosity
      float m_NN=PHASECTE[pp1].m_NN; float n_NN=PHASECTE[pp1].n_NN; float tau_yield=PHASECTE[pp1].tau_yield; float visco_NN=PHASECTE[pp1].visco;
      KerGetEta_Effective(pp1,tau_yield,D_tensor_magn,visco_NN,m_NN,n_NN,visco_etap1);

      //-Stores results.
      if(tvisco!=VISCO_Artificial) {
        //time step restriction
        if(visco_etap1>viscetadt[p1])viscetadt[p1]=visco_etap1; //no visceta necessary here
        //save deformation tensor
        float2 rg;
        rg=d_tensorff[p1*3];  rg=make_float2(rg.x+dtsrp1_xx_xy.x,rg.y+dtsrp1_xx_xy.y);  d_tensorff[p1*3]=rg;
        rg=d_tensorff[p1*3+1];  rg=make_float2(rg.x+dtsrp1_xz_yy.x,rg.y+dtsrp1_xz_yy.y);  d_tensorff[p1*3+1]=rg;
        rg=d_tensorff[p1*3+2];  rg=make_float2(rg.x+dtsrp1_yz_zz.x,rg.y+dtsrp1_yz_zz.y);  d_tensorff[p1*3+2]=rg;
        visco_eta[p1]=visco_etap1;
      }
    //auxnn[p1] = visco_etap1; // to be used if an auxilary is needed.
    }
  }
}

//------------------------------------------------------------------------------
/// Interaction of a particle with a set of particles. (Fluid/Float-Fluid/Float/Bound)
/// Realiza la interaccion de una particula con un conjunto de ellas. (Fluid/Float-Fluid/Float/Bound)
//------------------------------------------------------------------------------
template<TpKernel tker,TpFtMode ftmode,TpVisco tvisco,TpDensity tdensity,bool shift,bool symm>
__device__ void KerInteractionForcesFluidBox_NN_SPH_PressGrad(bool boundp2,unsigned p1
  ,const unsigned &pini,const unsigned &pfin
  ,const float *ftomassp
  ,const float4 *poscell
  ,float* distance_ratio, const float &chi_p1
  ,const float4 *velrhop,const typecode *code,const unsigned *idp
  ,float massp2,const typecode pp1,bool ftp1
  ,const float4 &pscellp1,const float4 &velrhop1
  ,float3 &grap1_xx_xy_xz,float3 &grap1_yx_yy_yz,float3 &grap1_zx_zy_zz
  ,float3 &acep1,float &arp1,float &visc,float &deltap1
  ,TpShifting shiftmode,float4 &shiftposfsp1,int Zhan_loop)
{ 
  for(int p2=pini; p2<pfin; p2++) {
    const float4 pscellp2=poscell[p2];
    float drx=pscellp1.x-pscellp2.x+CTE.poscellsize*(CEL_GetX(__float_as_int(pscellp1.w))-CEL_GetX(__float_as_int(pscellp2.w)));
    float dry=pscellp1.y-pscellp2.y+CTE.poscellsize*(CEL_GetY(__float_as_int(pscellp1.w))-CEL_GetY(__float_as_int(pscellp2.w)));
    float drz=pscellp1.z-pscellp2.z+CTE.poscellsize*(CEL_GetZ(__float_as_int(pscellp1.w))-CEL_GetZ(__float_as_int(pscellp2.w)));
    if(symm)dry=pscellp1.y+pscellp2.y+CTE.poscellsize*CEL_GetY(__float_as_int(pscellp2.w)); //<vs_syymmetry>
    const float rr2=drx*drx+dry*dry+drz*drz;
    if(rr2<=CTE.kernelsize2 && rr2>=ALMOSTZERO) {
      //-Computes kernel.
      const float fac=cufsph::GetKernel_Fac<tker>(rr2);
      const float frx=fac*drx,fry=fac*dry,frz=fac*drz; //-Gradients.

      //-Obtains mass of particle p2 for NN and if any floating bodies exist.
      const typecode cod=code[p2];
      const typecode pp2=(boundp2 ? pp1 : CODE_GetTypeValue(cod)); //<vs_non-Newtonian>
      float massp2;
      //Note if you masses are very different more than a ratio of 1.3 then: massp2 = (boundp2 ? PHASEARRAY[pp1].mass : PHASEARRAY[pp2].mass);
      if(tvisco==VISCO_Hypoplasticity){
        massp2=(boundp2 ? CTE.massb : PHASEHYPO[pp2].mass); 
      }else if(tvisco==VISCO_Elasticity){
        massp2=(boundp2 ? CTE.massb : PHASEELASTIC[pp2].mass); 
      }else massp2=(boundp2 ? CTE.massb : PHASEARRAY[pp2].mass);

      //-Obtiene masa de particula p2 en caso de existir floatings.
      bool ftp2=false;        //-Indicates if it is floating. | Indica si es floating.
      float ftmassp2;						//-Contains mass of floating body or massf if fluid. | Contiene masa de particula floating o massp2 si es bound o fluid.
      bool compute=true;			//-Deactivated when DEM is used and is float-float or float-bound. | Se desactiva cuando se usa DEM y es float-float o float-bound.
      if(USE_FLOATING) {
        const typecode cod=code[p2];
        ftp2=CODE_IsFloating(cod);
        ftmassp2=(ftp2 ? ftomassp[CODE_GetTypeValue(cod)] : massp2);
#ifdef DELTA_HEAVYFLOATING
        if(ftp2 && tdensity==DDT_DDT && ftmassp2<=(massp2*1.2f))deltap1=FLT_MAX;
#else
        if(ftp2 && tdensity==DDT_DDT)deltap1=FLT_MAX;
#endif
        if(ftp2 && shift && shiftmode==SHIFT_NoBound)shiftposfsp1.x=FLT_MAX; //-Cancels shifting with floating bodies. | Con floatings anula shifting.
        compute=!(USE_FTEXTERNAL && ftp1&&(boundp2||ftp2)); //-Deactivated when DEM or Chrono is used and is float-float or float-bound. | Se desactiva cuando se usa DEM o Chrono y es float-float o float-bound.
      }
      float4 velrhop2=velrhop[p2];
      if(symm)velrhop2.y=-velrhop2.y; //<vs_syymmetry>

      //===== Aceleration ===== 
      //if(compute) {
      //  if ((tvisco != VISCO_Hypoplasticity) & (tvisco != VISCO_Elasticity)){ ///pressure is not independent in hypoplastic model, burried in full stress tensor
      //    const float pressp2=cufsph::ComputePressCte_NN(velrhop2.w,PHASEARRAY[pp2].rho,PHASEARRAY[pp2].CteB,PHASEARRAY[pp2].Gamma,PHASEARRAY[pp2].Cs0,cod);
      //    const float prs=(pressp1+pressp2)/(velrhop1.w*velrhop2.w)+(tker==KERNEL_Cubic ? cufsph::GetKernelCubic_Tensil(rr2,velrhop1.w,pressp1,velrhop2.w,pressp2) : 0);
      //    const float p_vpm=-prs*(USE_FLOATING ? ftmassp2 : massp2);
      //    acep1.x+=p_vpm*frx; acep1.y+=p_vpm*fry; acep1.z+=p_vpm*frz;
      //  }
      //}

      //-Density derivative.
      float dvx=velrhop1.x-velrhop2.x,dvy=velrhop1.y-velrhop2.y,dvz=velrhop1.z-velrhop2.z;
      if(compute && tvisco != VISCO_Hypoplasticity && tvisco != VISCO_Elasticity) arp1+=(USE_FLOATING ? ftmassp2 : massp2)*(dvx*frx+dvy*fry+dvz*frz)*(velrhop1.w/velrhop2.w);
      // In hypo model, we used void ratio to follow the conservation of mass. density is just derived from void ratio.
      float cbar = 0.0;
      if(tvisco==VISCO_Hypoplasticity) {cbar=PHASEHYPO[pp2].Cs0;}
      else if(tvisco==VISCO_Elasticity) {cbar=PHASEELASTIC[pp2].Cs0;}
      else {cbar=PHASEARRAY[pp2].Cs0;}
      const float dot3=(tdensity!=DDT_None||shift ? drx*frx+dry*fry+drz*frz : 0);
      //-Density derivative (DeltaSPH Molteni).
      if(tdensity==DDT_DDT && deltap1!=FLT_MAX) {
        const float rhop1over2=velrhop1.w/velrhop2.w;
        const float visc_densi=CTE.ddtkh*cbar*(rhop1over2-1.f)/(rr2+CTE.eta2);
        const float delta=(pp1==pp2 ? visc_densi*dot3*(USE_FLOATING ? ftmassp2 : massp2) : 0); //<vs_non-Newtonian>
        //deltap1=(boundp2? FLT_MAX: deltap1+delta);
        deltap1=(boundp2 && CTE.tboundary==BC_DBC ? FLT_MAX : deltap1+delta);
      }
      //-Density Diffusion Term (Fourtakas et al 2019). //<vs_dtt2_ini>
      if((tdensity==DDT_DDT2||(tdensity==DDT_DDT2Full&&!boundp2))&&deltap1!=FLT_MAX&&!ftp2) {
        const float rh=1.f+CTE.ddtgz*drz;
        const float drhop=CTE.rhopzero*pow(rh,1.f/CTE.gamma)-CTE.rhopzero;
        const float visc_densi=CTE.ddtkh*cbar*((velrhop2.w-velrhop1.w)-drhop)/(rr2+CTE.eta2);
        const float delta=(pp1==pp2 ? visc_densi*dot3*massp2/velrhop2.w : 0); //<vs_non-Newtonian>
        deltap1=(boundp2 ? FLT_MAX : deltap1-delta);
      } //<vs_dtt2_end>		

      //-Shifting correction.
      if(shift && shiftposfsp1.x!=FLT_MAX) {
        bool heavyphase;
        if(tvisco==VISCO_Hypoplasticity){
          heavyphase=(PHASEHYPO[pp1].mass>PHASEHYPO[pp2].mass && pp1!=pp2 ? true : false);
        }else if(tvisco==VISCO_Elasticity){
          heavyphase=(PHASEELASTIC[pp1].mass>PHASEELASTIC[pp2].mass && pp1!=pp2 ? true : false);
        }else{
          heavyphase=(PHASEARRAY[pp1].mass>PHASEARRAY[pp2].mass && pp1!=pp2 ? true : false);
        }
        const float massrhop=(USE_FLOATING ? ftmassp2 : massp2)/velrhop2.w;
        const bool noshift=(boundp2&&(shiftmode==SHIFT_NoBound||(shiftmode==SHIFT_NoFixed && CODE_IsFixed(code[p2]))));
        shiftposfsp1.x=(noshift ? FLT_MAX : (heavyphase ? 0 : shiftposfsp1.x+massrhop*frx)); //-Removes shifting for the boundaries. | Con boundary anula shifting.
        shiftposfsp1.y+=(heavyphase ? 0 : massrhop*fry);
        shiftposfsp1.z+=(heavyphase ? 0 : massrhop*frz);
        shiftposfsp1.w-=(heavyphase ? 0 : massrhop*dot3);
      }
      //===== Viscosity ===== 
      if(compute) {
        const float dot=drx*dvx+dry*dvy+drz*dvz;
        //if (p1 == 113 && p2 == 114){
        //  printf("%s \n","In pressgrad");
        //  printf("drx dry drz dvx dvy dvz dot: %f %f %f %f %f %f %f\n",drx,dry,drz,dvx,dvy,dvz,dot);
        //}
        const float dot_rr2=dot/(rr2+CTE.eta2);
        visc=max(dot_rr2,visc);  //ViscDt=max(dot/(rr2+Eta2),ViscDt);
        if(tvisco!=VISCO_Artificial) { //&& !boundp2
          // vel gradients 
          // if(boundp2 & tvisco != VISCO_Hypoplasticity) { //jinw@ need come back and to check the boundary
          if (boundp2 & (tvisco != VISCO_Hypoplasticity) && (tvisco != VISCO_Elasticity)){
            dvx=2.f*velrhop1.x; dvy=2.f*velrhop1.y; dvz=2.f*velrhop1.z;  //fomraly I should use the moving BC vel as ug=2ub-uf
            
            if(tvisco != VISCO_Hypoplasticity){
              dvx=2.f*velrhop1.x; dvy=2.f*velrhop1.y; dvz=2.f*velrhop1.z;  //fomraly I should use the moving BC vel as ug=2ub-uf
            }else{
              const float chi_p2=distance_ratio[p2];
              float coef=1;
              if (chi_p1 >= 0.5 & chi_p2 >= 0.5){
                coef=min(1.5, (chi_p2*2-1)/(chi_p1*2-1)+1);
              }
              // printf("particle p1=%d, coef=%f, chi_p1=%f, chi_p2=%f\n",p1, coef,chi_p1,chi_p2);
              dvx=coef*dvx; dvy=coef*dvy; dvz=coef*dvz;  //fomraly I should use the moving BC vel as ug=2ub-uf
            }
            
          }
          float vel_check = sqrt(pow(velrhop2.x,2)+pow(velrhop2.y,2)+pow(velrhop2.z,2))/sqrt(pow(velrhop1.x,2)+pow(velrhop1.y,2)+pow(velrhop1.z,2));
          if ((vel_check<5) && (Zhan_loop==0) || (!boundp2)){
            GetVelocityGradients_SPH_tsym(massp2,velrhop2,dvx,dvy,dvz,frx,fry,frz,grap1_xx_xy_xz,grap1_yx_yy_yz,grap1_zx_zy_zz);
          }
        }//jinw@-Artificial viscosity.
        if((tvisco == VISCO_Hypoplasticity) || (tvisco == VISCO_Elasticity)) {
          // const float cbar= max(PHASEHYPO[pp1].Cs0,PHASEHYPO[pp2].Cs0);
          float cbar = 0.0;
          float visco_NN = 0.0;
          if (tvisco == VISCO_Hypoplasticity){
            cbar = PHASEHYPO[pp2].Cs0;
            visco_NN= PHASEHYPO[pp2].visco;
          }else{
            cbar = PHASEELASTIC[pp2].Cs0;
            visco_NN= PHASEELASTIC[pp2].visco; 
          }
          if(dot<0) {
            const float amubar=CTE.kernelh*dot_rr2;  //amubar=CTE.h*dot/(rr2+CTE.eta2);
            const float robar=(velrhop1.w+velrhop2.w)*0.5f;
            const float pi_visc=(-visco_NN*cbar*amubar/robar)*massp2;
            acep1.x-=pi_visc*frx; acep1.y-=pi_visc*fry; acep1.z-=pi_visc*frz;
          }
        }
      }
    }
  }
  //if (p1 == 113){
  //  printf("acep1: %f %f %f\n",acep1.x, acep1.y, acep1.z);
  //} 
}


//------------------------------------------------------------------------------
/// Interaction between particles for non-Newtonian models using the SPH approach. Fluid/Float-Fluid/Float or Fluid/Float-Bound.
/// Includes pressure calculations, velocity gradients and normal/DEM floating bodies.
///
/// Realiza interaccion entre particulas para modelos no-Newtonianos que utilizan el enfoque de la SPH. Fluid/Float-Fluid/Float or Fluid/Float-Bound
/// Incluye visco artificial/laminar y floatings normales/dem.
//------------------------------------------------------------------------------
template<TpKernel tker,TpFtMode ftmode,TpVisco tvisco,TpDensity tdensity,bool shift,bool symm>
__global__ void KerInteractionForcesFluid_NN_SPH_PressGrad(unsigned n,unsigned pinit
  ,int scelldiv,int4 nc,int3 cellzero,const int2 *begincell,unsigned cellfluid,const unsigned *dcell
  ,const float *ftomassp,float3 *gradvelff
  ,const float4 *poscell
  ,const float4 *velrhop,const typecode *code,const unsigned *idp
  ,float *viscdt,float *ar,float3 *ace,float *delta
  ,TpShifting shiftmode,float4 *shiftposfs
  ,float* distance_ratio
  ,float ViscDtMax, float MaxVel,int Zhan_loop)
{
  const unsigned p=blockIdx.x*blockDim.x+threadIdx.x; //-Number of particle.
  if(p<n) {
    unsigned p1=p+pinit;      //-Number of particle.
    float visc=0,arp1=0,deltap1=0;
    float3 acep1=make_float3(0,0,0);
    const float chi_p1=distance_ratio[p1];

    //-Variables for Shifting.
    float4 shiftposfsp1;
    if(shift)shiftposfsp1=shiftposfs[p1];

    //-Obtains data of particle p1 in case there are floating bodies.
    //-Obtiene datos de particula p1 en caso de existir floatings.
    bool ftp1;       //-Indicates if it is floating. | Indica si es floating.
    const typecode cod=code[p1];
    if(USE_FLOATING) {
      ftp1=CODE_IsFloating(cod);
      if(ftp1 && tdensity!=DDT_None)deltap1=FLT_MAX; //-DDT is not applied to floating particles.
      if(ftp1 && shift)shiftposfsp1.x=FLT_MAX; //-Shifting is not calculated for floating bodies. | Para floatings no se calcula shifting.
    }

    //-Obtains basic data of particle p1.		
    const float4 pscellp1=poscell[p1];
    const float4 velrhop1=velrhop[p1];
    //<vs_non-Newtonian>
    const typecode pp1=CODE_GetTypeValue(cod);

    //Obtain pressure using state equation
    //float pressp1=0;
    //if((tvisco!=VISCO_Hypoplasticity) && (tvisco!=VISCO_Elasticity)){
    //  pressp1=cufsph::ComputePressCte_NN(velrhop1.w,PHASEARRAY[pp1].rho,PHASEARRAY[pp1].CteB,PHASEARRAY[pp1].Gamma,PHASEARRAY[pp1].Cs0,cod);
    //}
    
    const bool rsymp1=(symm && CEL_GetPartY(__float_as_uint(pscellp1.w))==0); //<vs_syymmetry>

    //-Variables for vel gradients
    float3 grap1_xx_xy_xz,grap1_yx_yy_yz,grap1_zx_zy_zz;
    if(tvisco!=VISCO_Artificial) {
      grap1_xx_xy_xz=make_float3(0,0,0);
      grap1_yx_yy_yz=make_float3(0,0,0);
      grap1_zx_zy_zz=make_float3(0,0,0);
    }

    //-Obtains neighborhood search limits.
    int ini1,fin1,ini2,fin2,ini3,fin3;
    cunsearch::InitCte(dcell[p1],scelldiv,nc,cellzero,ini1,fin1,ini2,fin2,ini3,fin3);

    //-Interaction with fluids.
    ini3+=cellfluid; fin3+=cellfluid;
    for(int c3=ini3; c3<fin3; c3+=nc.w)for(int c2=ini2; c2<fin2; c2+=nc.x) {
      unsigned pini,pfin=0; cunsearch::ParticleRange(c2,c3,ini1,fin1,begincell,pini,pfin);
      if(pfin) {
        KerInteractionForcesFluidBox_NN_SPH_PressGrad<tker,ftmode,tvisco,tdensity,shift,false>(false,p1,pini,pfin,ftomassp,poscell,distance_ratio,chi_p1,velrhop,code,idp,CTE.massf,pp1,ftp1,pscellp1,velrhop1,grap1_xx_xy_xz,grap1_yx_yy_yz,grap1_zx_zy_zz,acep1,arp1,visc,deltap1,shiftmode,shiftposfsp1,Zhan_loop);
        if(symm && rsymp1)	KerInteractionForcesFluidBox_NN_SPH_PressGrad<tker,ftmode,tvisco,tdensity,shift,true >(false,p1,pini,pfin,ftomassp,poscell,distance_ratio,chi_p1,velrhop,code,idp,CTE.massf,pp1,ftp1,pscellp1,velrhop1,grap1_xx_xy_xz,grap1_yx_yy_yz,grap1_zx_zy_zz,acep1,arp1,visc,deltap1,shiftmode,shiftposfsp1,Zhan_loop); //<vs_syymmetry>
      }
    }
    if(Zhan_loop!=1){
      //-Interaction with boundaries.
      ini3-=cellfluid; fin3-=cellfluid;
      for(int c3=ini3; c3<fin3; c3+=nc.w)for(int c2=ini2; c2<fin2; c2+=nc.x) {
        unsigned pini,pfin=0; cunsearch::ParticleRange(c2,c3,ini1,fin1,begincell,pini,pfin);
        if(pfin) {
          KerInteractionForcesFluidBox_NN_SPH_PressGrad<tker,ftmode,tvisco,tdensity,shift,false>(true,p1,pini,pfin,ftomassp,poscell,distance_ratio,chi_p1,velrhop,code,idp,CTE.massb,pp1,ftp1,pscellp1,velrhop1,grap1_xx_xy_xz,grap1_yx_yy_yz,grap1_zx_zy_zz,acep1,arp1,visc,deltap1,shiftmode,shiftposfsp1,Zhan_loop);
          if(symm && rsymp1)	KerInteractionForcesFluidBox_NN_SPH_PressGrad<tker,ftmode,tvisco,tdensity,shift,true >(true,p1,pini,pfin,ftomassp,poscell,distance_ratio,chi_p1,velrhop,code,idp,CTE.massb,pp1,ftp1,pscellp1,velrhop1,grap1_xx_xy_xz,grap1_yx_yy_yz,grap1_zx_zy_zz,acep1,arp1,visc,deltap1,shiftmode,shiftposfsp1,Zhan_loop); //<vs_syymmetry>
        }
      }
    }
    //-Stores results.
    if(shift||arp1||acep1.x||acep1.y||acep1.z||visc) {
      if(tdensity!=DDT_None) {
        if(delta) {
          const float rdelta=delta[p1];
          delta[p1]=(rdelta==FLT_MAX||deltap1==FLT_MAX ? FLT_MAX : rdelta+deltap1);
        }
        else if(deltap1!=FLT_MAX)arp1+=deltap1;
      }
      ar[p1]+=arp1;
      if(Zhan_loop==0 || Zhan_loop==1) {
        float3 r=ace[p1]; r.x+=acep1.x; r.y+=acep1.y; r.z+=acep1.z; ace[p1]=r;
      }
      ViscDtMax = max(ViscDtMax,visc);
      MaxVel = max(MaxVel, velrhop1.x*velrhop1.x+velrhop1.y*velrhop1.y+velrhop1.z*velrhop1.z);
      if(visc>viscdt[p1])viscdt[p1]=visc;
      //jinw@ tmatrix3f is casted into 3*float3
      if(tvisco!=VISCO_Artificial) {
        float3 rg;
        rg=gradvelff[p1*3];		 rg=make_float3(rg.x+grap1_xx_xy_xz.x, rg.y+grap1_xx_xy_xz.y, rg.z+grap1_xx_xy_xz.z);  gradvelff[p1*3]=rg;
        rg=gradvelff[p1*3+1];  rg=make_float3(rg.x+grap1_yx_yy_yz.x, rg.y+grap1_yx_yy_yz.y, rg.z+grap1_yx_yy_yz.z);  gradvelff[p1*3+1]=rg;
        rg=gradvelff[p1*3+2];  rg=make_float3(rg.x+grap1_zx_zy_zz.x, rg.y+grap1_zx_zy_zz.y, rg.z+grap1_zx_zy_zz.z);  gradvelff[p1*3+2]=rg;
        // if (grap1_xx_xy_xz.x==nan) || (grap1_xx_xy_xz.y==nan) || (grap1_xx_xy_xz.z==nan)
      }
      if(shift)shiftposfs[p1]=shiftposfsp1;
      //auxnn[p1] = visco_etap1; // to be used if an auxilary is needed.
    }
  }
  //printf("ace[pinit].x:%f \n",ace[pinit].x);
}


//==============================================================================
template<TpKernel tker,bool symm>
__device__ void KerInteractionFluid_weight_accumulation(const unsigned &pini,const unsigned &pfin
  ,const float4 *poscell,const float4 &pscellp1
  ,float &weight_acc)
{
  for(int p2=pini; p2<pfin; p2++) {
    const float4 pscellp2=poscell[p2];
    float drx=pscellp1.x-pscellp2.x+CTE.poscellsize*(CEL_GetX(__float_as_int(pscellp1.w))-CEL_GetX(__float_as_int(pscellp2.w)));
    float dry=pscellp1.y-pscellp2.y+CTE.poscellsize*(CEL_GetY(__float_as_int(pscellp1.w))-CEL_GetY(__float_as_int(pscellp2.w)));
    float drz=pscellp1.z-pscellp2.z+CTE.poscellsize*(CEL_GetZ(__float_as_int(pscellp1.w))-CEL_GetZ(__float_as_int(pscellp2.w)));
    if(symm)dry=pscellp1.y+pscellp2.y+CTE.poscellsize*CEL_GetY(__float_as_int(pscellp2.w)); //<vs_syymmetry>
    const float rr2=drx*drx+dry*dry+drz*drz;
    if(rr2<=CTE.kernelsize2 && rr2>=ALMOSTZERO) {
      //-Computes kernel.
      const float fac=cufsph::GetKernel_Fac<tker>(rr2);
      weight_acc+=fac;
    }
  }
}
//------------------------------------------------------------------------------
/// Calculate distance for boundary condition enforcement
//------------------------------------------------------------------------------
template<TpKernel tker,TpFtMode ftmode,TpVisco tvisco,bool symm>
__global__ void KerInteractionForcesFluid_Boundary_Distance(unsigned n,unsigned pinit
  ,int scelldiv,int4 nc,int3 cellzero,const int2 *begincell,unsigned cellfluid,const unsigned *dcell
  ,const float4 *poscell, float* distance_ratio)
{
  const unsigned p=blockIdx.x*blockDim.x+threadIdx.x; //-Number of particle.
  if(p<n) {
    unsigned p1=p+pinit;      //-Number of particle.
    float weight_fluid=0,weight_bound=0;
    const float4 pscellp1=poscell[p1];
    // const bool rsymp1=(symm && CEL_GetPartY(__float_as_uint(pscellp1.w))==0); //<vs_syymmetry>

    //-Obtains neighborhood search limits.
    int ini1,fin1,ini2,fin2,ini3,fin3;
    cunsearch::InitCte(dcell[p1],scelldiv,nc,cellzero,ini1,fin1,ini2,fin2,ini3,fin3);

    //-Interaction with fluids.
    ini3+=cellfluid; fin3+=cellfluid;
    for(int c3=ini3; c3<fin3; c3+=nc.w)for(int c2=ini2; c2<fin2; c2+=nc.x) {
      unsigned pini,pfin=0; cunsearch::ParticleRange(c2,c3,ini1,fin1,begincell,pini,pfin);
      if(pfin) {
        KerInteractionFluid_weight_accumulation<tker,symm>(pini,pfin,poscell,pscellp1,weight_fluid);
      }
    }
    //-Interaction with boundaries.
    ini3-=cellfluid; fin3-=cellfluid;
    for(int c3=ini3; c3<fin3; c3+=nc.w)for(int c2=ini2; c2<fin2; c2+=nc.x) {
      unsigned pini,pfin=0; cunsearch::ParticleRange(c2,c3,ini1,fin1,begincell,pini,pfin);
      if(pfin) {
        KerInteractionFluid_weight_accumulation<tker,symm>(pini,pfin,poscell,pscellp1,weight_bound);
      }
    }

    const float chi = weight_fluid/(weight_bound + weight_fluid);
    //-Stores results.
    distance_ratio[p1]=chi;
    // if(chi<1) {
    //   printf("particle id=%d, distance_ratio = %f \n", p1, distance_ratio[p1]);
    // }
  }
}

//------------------------------------------------------------------------------
/// Particle interaction for non-Newtonian models. Bound-Fluid/Float 
/// Realiza interaccion entre particulas para modelos no-Newtonianos. Bound-Fluid/Float
//------------------------------------------------------------------------------
template<TpKernel tker,TpFtMode ftmode, TpVisco tvisco, bool symm>
__global__ void KerInteractionForcesBoundary_Fluid_Distance(unsigned n,unsigned pinit
  ,int scelldiv,int4 nc,int3 cellzero,const int2 *begincell,unsigned cellfluid,const unsigned *dcell
  ,const float4 *poscell, float* distance_ratio)
{
  const unsigned p=blockIdx.x*blockDim.x+threadIdx.x; //-Number of thread.
  if(p<n) {
    const unsigned p1=p+pinit;      //-Number of particle.
    float weight_fluid=0,weight_bound=0;
    //-Loads particle p1 data.
    const float4 pscellp1=poscell[p1];

    //-Obtains neighborhood search limits.
    int ini1,fin1,ini2,fin2,ini3,fin3;
    cunsearch::InitCte(dcell[p1],scelldiv,nc,cellzero,ini1,fin1,ini2,fin2,ini3,fin3);

    //-Interaction with fluids.
    ini3+=cellfluid; fin3+=cellfluid;
    for(int c3=ini3; c3<fin3; c3+=nc.w)for(int c2=ini2; c2<fin2; c2+=nc.x) {
      unsigned pini,pfin=0; cunsearch::ParticleRange(c2,c3,ini1,fin1,begincell,pini,pfin);
      if(pfin) {
        KerInteractionFluid_weight_accumulation<tker,symm>(pini,pfin,poscell,pscellp1,weight_fluid);
      }
    }
    //-Interaction with boundaries.
    ini3-=cellfluid; fin3-=cellfluid;
    for(int c3=ini3; c3<fin3; c3+=nc.w)for(int c2=ini2; c2<fin2; c2+=nc.x) {
      unsigned pini,pfin=0; cunsearch::ParticleRange(c2,c3,ini1,fin1,begincell,pini,pfin);
      if(pfin) {
        KerInteractionFluid_weight_accumulation<tker,symm>(pini,pfin,poscell,pscellp1,weight_bound);
      }
    }
    const float chi = weight_bound/(weight_bound + weight_fluid);
    //-Stores results.
    distance_ratio[p1]=chi;
    // if(chi<1) {
    //   printf("particle id=%d, distance_ratio = %f \n", p1, distance_ratio[p1]);
    // }
  }
}

//==============================================================================
/// Interaction for the force computation using the SPH approach.
/// Interaccion para el calculo de fuerzas que utilizan el enfoque de la SPH .
//==============================================================================
template<TpKernel tker,TpFtMode ftmode,TpVisco tvisco,TpDensity tdensity,bool shift,TpSlipMode slipmode>
void Interaction_ForcesGpuT_NN_SPH(const StInterParmsg &t,double time_inc,int &Zhan_loop)
{
  bool stop=false;
  //-Collects kernel information.

#ifndef DISABLE_BSMODES
  if(t.kerinfo) {
    cusph::Interaction_ForcesT_KerInfo<tker,ftmode,true,tdensity,shift,false>(t.kerinfo);
    return;
  }
#endif
  const StDivDataGpu &dvd=t.divdatag;
  // const int2* beginendcell=dvd.beginendcell;
  dim3 sgridf=GetSimpleGridSize(t.fluidnum,t.bsfluid);
  dim3 sgridb=GetSimpleGridSize(t.boundnum,t.bsbound);
  double dt = 0;
  
  //printf("At JSphGpu_NN_ker.cu::Interaction_ForcesGpuT_NN_SPH");
  //printf("t.velrhop: %f %f %f %f", t.velrhop[t.boundnum].x,t.velrhop[t.boundnum].y,t.velrhop[t.boundnum].z,t.velrhop[t.boundnum].w);
  //printf("t.BoundNormalg: %f %f %f", t.BoundNormalg[t.boundnum-1].x,t.BoundNormalg[t.boundnum-1].y,t.BoundNormalg[t.boundnum-1].z);
  //printf("t.void_ratio: %f", t.void_ratio[t.boundnum]);

  //-Interaction Fluid-Fluid & Fluid-Bound.
  if(t.fluidnum) { 
    if(!t.symmetry){ //<not symmetic>
      float ViscDtMax=0;
      float MaxVel=0;
      if(Zhan_loop != 2){
        KerInteractionForcesFluid_NN_SPH_PressGrad<tker,ftmode,tvisco,tdensity,shift,false ><<<sgridf,t.bsfluid,0,t.stm>>>
          (t.fluidnum,t.fluidini,dvd.scelldiv,dvd.nc,dvd.cellzero,dvd.beginendcell,dvd.cellfluid,t.dcell
            ,t.ftomassp,(float3*)t.gradvel,t.poscell,t.velrhop,t.code,t.idp
            ,t.viscdt,t.ar,t.ace,t.delta,t.shiftmode,t.shiftposfs,t.distance_ratio,ViscDtMax,MaxVel,Zhan_loop);
        //if((tvisco!=VISCO_Artificial) & (tvisco!=VISCO_Hypoplasticity) & (tvisco!=VISCO_Elasticity) ) {
        //  KerInteractionForcesFluid_NN_SPH_Visco_eta<ftmode,tvisco,false ><<<sgridf,t.bsfluid,0,t.stm>>>
        //    (t.fluidnum,t.fluidini,t.viscob,t.visco_eta,t.velrhop,dvd.scelldiv,dvd.nc,dvd.cellzero,dvd.beginendcell,dvd.cellfluid,t.dcell
        //      ,(float2*)t.d_tensor,t.w_tensor,(float3*)t.gradvel,t.code,t.idp
        //      ,t.viscetadt);
        //}
      }
      //choice of visc formulation
      if((tvisco!=VISCO_ConstEq) & (tvisco!=VISCO_Hypoplasticity) & (tvisco!=VISCO_Elasticity)){
        KerInteractionForcesFluid_NN_SPH_Morris<tker,ftmode,tvisco,false ><<<sgridf,t.bsfluid,0,t.stm>>>
        (t.fluidnum,t.fluidini,t.viscob,t.viscof,t.visco_eta,dvd.scelldiv,dvd.nc,dvd.cellzero,dvd.beginendcell,dvd.cellfluid,t.dcell
          ,t.ftomassp,t.auxnn,t.poscell,t.velrhop,t.code,t.idp
          ,t.ace);
      }else{
        if (time_inc==0){
          dt = double(CTE.cfl_number * ((CTE.kernelh) / (max(float(CTE.cs0), t.VelMax * 10.) + (CTE.kernelh)*ViscDtMax)));
        }else{
          dt = time_inc;
        }
        if(Zhan_loop!=2){
       //   // Build stress tensor
        KerInteractionForcesFluid_NN_SPH_Visco_Stress_tensor<tker,ftmode,tvisco,false ><<<sgridf,t.bsfluid,0,t.stm>>>
           (t.fluidnum,t.fluidini,t.visco_eta,dvd.scelldiv,dvd.nc,dvd.cellzero,dvd.beginendcell,dvd.cellfluid,t.dcell
             ,t.ftomassp,(float2*)t.tau,t.poscell,t.velrhop,t.void_ratio,t.code,(float3*)t.gradvel,dt,stop,t.regularize);
       //   //Get stresses. If dummay particle method, f-f &f-b; if Zhan's friction method, only f-f
        KerInteractionForcesFluid_NN_SPH_ConsEq<tker,ftmode,tvisco,false ><<<sgridf,t.bsfluid,0,t.stm>>>
            (t.fluidnum,t.fluidini,t.visco_eta,dvd.scelldiv,dvd.nc,dvd.cellzero,dvd.beginendcell,dvd.cellfluid,t.dcell
              ,t.ftomassp,(float2*)t.tau,t.poscell,t.velrhop,t.code,t.ace,Zhan_loop,t.regularize);
        }
       // // When Zhan_loop = 1: prediction step without boundary effect. Zhan_loop = 2: correction considering only the boundary effect.  
        if (Zhan_loop == 2){           
          KerInteractionForcesFluid_NN_SPH_ConsEq_Zhan_bound<tker,ftmode,tvisco,slipmode,false ><<<sgridf,t.bsfluid,0,t.stm>>>
            (t.fluidnum,t.fluidini,t.visco_eta,dvd.scelldiv,dvd.nc,dvd.cellzero,dvd.beginendcell,t.dcell
              ,t.ftomassp,(float2*)t.tau,t.poscell,t.velrhop,t.code,t.ace,t.BoundNormalg,t.BoundCornerg,t.Forceg,dt);
        }
      } 
    }
    /*
    else {//<vs_syymmetry_end> symmetry pennding implementing
      KerInteractionForcesFluid_Boundary_Distance<tker,ftmode,tvisco,false ><<<sgridf,t.bsfluid,0,t.stm>>>
      (t.fluidnum,t.fluidini,dvd.scelldiv,dvd.nc,dvd.cellzero,dvd.beginendcell,dvd.cellfluid,t.dcell
      ,t.poscell,t.distance_ratio);
      KerInteractionForcesBoundary_Fluid_Distance<tker,ftmode,tvisco,false ><<<sgridb,t.bsbound,0,t.stm>>>
      (t.boundnum,t.boundini,dvd.scelldiv,dvd.nc,dvd.cellzero,dvd.beginendcell,dvd.cellfluid,t.dcell
      ,t.poscell,t.distance_ratio);
      float ViscDtMax=0;
      float MaxVel=0;
      ///obtian the strain gradient full tensor			
      KerInteractionForcesFluid_NN_SPH_PressGrad<tker,ftmode,tvisco,tdensity,shift,false><<<sgridf,t.bsfluid,0,t.stm>>>
        (t.fluidnum,t.fluidini,dvd.scelldiv,dvd.nc,dvd.cellzero,dvd.beginendcell,dvd.cellfluid,t.dcell
          ,t.ftomassp,(float3*)t.gradvel,t.poscell,t.velrhop,t.code,t.idp
          ,t.viscdt,t.ar,t.ace,t.delta,t.shiftmode,t.shiftposfs,t.distance_ratio,ViscDtMax,MaxVel);
      //Build strain rate tensor and spin tensor and compute eta_visco 
      if(tvisco!=VISCO_Hypoplasticity )KerInteractionForcesFluid_NN_SPH_Visco_eta<ftmode,tvisco,false ><<<sgridf,t.bsfluid,0,t.stm>>>
        (t.fluidnum,t.fluidini,t.viscob,t.visco_eta,t.velrhop,dvd.scelldiv,dvd.nc,dvd.cellzero,dvd.beginendcell,dvd.cellfluid,t.dcell
          ,(float2*)t.d_tensor,t.w_tensor,(float3*)t.gradvel,t.code,t.idp
          ,t.viscetadt);
      //choice of visc formulation
      if(tvisco!=VISCO_ConstEq & tvisco!=VISCO_Hypoplasticity ){
        KerInteractionForcesFluid_NN_SPH_Morris<tker,ftmode,tvisco,false ><<<sgridf,t.bsfluid,0,t.stm>>>
        (t.fluidnum,t.fluidini,t.viscob,t.viscof,t.visco_eta,dvd.scelldiv,dvd.nc,dvd.cellzero,dvd.beginendcell,dvd.cellfluid,t.dcell
          ,t.ftomassp,t.auxnn,t.poscell,t.velrhop,t.code,t.idp
          ,t.ace);
      }else{
        if (dt==0){
          dt = double(CTE.cfl_number * (CTE.kernelh / (max(CTE.cs0, sqrt(MaxVel) * 10.) + CTE.kernelh*ViscDtMax)));
        }
        // if (dt==0) printf("dt =========0\n");
        // Build stress tensor				
        KerInteractionForcesFluid_NN_SPH_Visco_Stress_tensor<ftmode,tvisco,false ><<<sgridf,t.bsfluid,0,t.stm>>>
          (t.fluidnum,t.fluidini,t.visco_eta,dvd.scelldiv,dvd.nc,dvd.cellzero,dvd.beginendcell,dvd.cellfluid,t.dcell
            ,t.ftomassp,(float2*)t.tau,(float2*)t.d_tensor,t.velrhop,t.void_ratio,t.w_tensor,t.auxnn,t.poscell,t.velrhop,t.code,t.idp,(float3*)t.gradvel,dt,stop);
        //Get stresses contribution to the force terms
        KerInteractionForcesFluid_NN_SPH_ConsEq<tker,ftmode,tvisco,false ><<<sgridf,t.bsfluid,0,t.stm>>>
          (t.fluidnum,t.fluidini,t.viscob,t.viscof,t.visco_eta,dvd.scelldiv,dvd.nc,dvd.cellzero,dvd.beginendcell,dvd.cellfluid,t.dcell
            ,t.ftomassp,(float2*)t.tau,t.auxnn,t.poscell,t.velrhop,t.code,t.idp
            ,t.ace,regularize);
      }
    }
    */
  } 
  if (stop){
    exit(EXIT_FAILURE);
  } 
  //-Interaction Boundary-Fluid.
  /*
  if(t.boundnum & (tvisco!=VISCO_Hypoplasticity) & (tvisco!=VISCO_Elasticity)) {
    const int2* beginendcellfluid=dvd.beginendcell+dvd.cellfluid;
    //printf("bsbound:%u\n",bsbound);
    if(t.symmetry) //<vs_syymmetry_ini>
      KerInteractionForcesBound_NN<tker,ftmode,tvisco,true ><<<sgridb,t.bsbound,0,t.stm>>>
      (t.boundnum,t.boundini,dvd.scelldiv,dvd.nc,dvd.cellzero,beginendcellfluid,t.dcell
        ,t.ftomassp,(float2*)t.tau,t.poscell,t.velrhop,t.code,t.idp,t.viscdt,t.ar,t.ace,dt);
    else //<vs_syymmetry_end>
    //jinw@need work on here for boundary implementation
      KerInteractionForcesBound_NN<tker,ftmode,tvisco,false><<<sgridb,t.bsbound,0,t.stm>>>
      (t.boundnum,t.boundini,dvd.scelldiv,dvd.nc,dvd.cellzero,beginendcellfluid,t.dcell
        ,t.ftomassp,(float2*)t.tau,t.poscell,t.velrhop,t.code,t.idp,t.viscdt,t.ar,t.ace,dt);
  }
  */
}
//======================END of SPH==============================================

//======================Start of non-Newtonian Templates=======================================
//Uncomment for fast compile 
//#define FAST_COMPILATION
template<TpKernel tker,TpFtMode ftmode,TpVisco tvisco,TpDensity tdensity,bool shift,TpSlipMode slipmode> void Interaction_ForcesGpuT_NN(const StInterParmsg &t,double time_inc, int Zhan_loop) {
#ifdef FAST_COMPILATION
  if(t.tvelgrad!=VELGRAD_FDA)throw "Extra SPH Gradients options are disabled for FastCompilation...";
  //Interaction_ForcesGpuT_NN_FDA	    < tker,ftmode,tvisco,tdensity,shift>(t,Zhan_loop);
#else	
  //if(t.tvelgrad==VELGRAD_FDA) Interaction_ForcesGpuT_NN_FDA	    < tker,ftmode,tvisco,tdensity,shift>(t,Zhan_loop);
  if(t.tvelgrad==VELGRAD_SPH)	Interaction_ForcesGpuT_NN_SPH		< tker,ftmode,tvisco,tdensity,shift,slipmode>(t,time_inc,Zhan_loop);
#endif
}
//==============================================================================
template<TpKernel tker,TpFtMode ftmode,TpVisco tvisco,TpDensity tdensity,bool shift> void Interaction_ForcesGpuT_NN_gt3(const StInterParmsg &t,double time_inc, int Zhan_loop) {
#ifdef FAST_COMPILATION
  if(t.slipmode!=SLIP_Vel0)throw "Zhan's slips are disabled for FastCompilation...";
  Interaction_ForcesGpuT_NN<tker,ftmode,tvisco,tdensity,shift,SLIP_Vel0>(t,time_inc,Zhan_loop);
#else
  if(t.slipmode==SLIP_NoSlip_Zhan)		      Interaction_ForcesGpuT_NN<tker,ftmode,tvisco,tdensity,shift,SLIP_NoSlip_Zhan>(t,time_inc,Zhan_loop);
  else if(t.slipmode==SLIP_FreeSlip_Zhan)	Interaction_ForcesGpuT_NN<tker,ftmode,tvisco,tdensity,shift,SLIP_FreeSlip_Zhan>(t,time_inc,Zhan_loop);
  else if(t.slipmode==SLIP_Friction_Zhan)	Interaction_ForcesGpuT_NN<tker,ftmode,tvisco,tdensity,shift,SLIP_Friction_Zhan>(t,time_inc,Zhan_loop);
  else if(t.slipmode==SLIP_Vel0)Interaction_ForcesGpuT_NN<tker,ftmode,tvisco,tdensity,shift,SLIP_Vel0>(t,time_inc,Zhan_loop);
  else throw "Other slip modes have not been implemented";
#endif
}

//==============================================================================
template<TpKernel tker,TpFtMode ftmode,TpVisco tvisco> void Interaction_ForcesNN_gt2(const StInterParmsg &t,double time_inc,int Zhan_loop) {
#ifdef FAST_COMPILATION
  if(!t.shiftmode||t.tdensity!=DDT_DDT2Full)throw "Shifting and extra DDT are disabled for FastCompilation...";
  Interaction_ForcesGpuT_NN_gt3<tker,ftmode,tvisco,DDT_DDT2Full,true>(t,time_inc,Zhan_loop);
#else
  if(t.shiftmode) {
    const bool shift=true;
    if(t.tdensity==DDT_None)    Interaction_ForcesGpuT_NN_gt3<tker,ftmode,tvisco,DDT_None,shift>(t,time_inc,Zhan_loop);
    if(t.tdensity==DDT_DDT)     Interaction_ForcesGpuT_NN_gt3<tker,ftmode,tvisco,DDT_DDT,shift>(t,time_inc,Zhan_loop);
    if(t.tdensity==DDT_DDT2)    Interaction_ForcesGpuT_NN_gt3<tker,ftmode,tvisco,DDT_DDT2,shift>(t,time_inc,Zhan_loop);  //<vs_dtt2>
    if(t.tdensity==DDT_DDT2Full)Interaction_ForcesGpuT_NN_gt3<tker,ftmode,tvisco,DDT_DDT2Full,shift>(t,time_inc,Zhan_loop);  //<vs_dtt2>
  }
  else {
    const bool shift=false;
    if(t.tdensity==DDT_None)    Interaction_ForcesGpuT_NN_gt3<tker,ftmode,tvisco,DDT_None,shift>(t,time_inc,Zhan_loop);
    if(t.tdensity==DDT_DDT)     Interaction_ForcesGpuT_NN_gt3<tker,ftmode,tvisco,DDT_DDT,shift>(t,time_inc,Zhan_loop);
    if(t.tdensity==DDT_DDT2)    Interaction_ForcesGpuT_NN_gt3<tker,ftmode,tvisco,DDT_DDT2,shift>(t,time_inc,Zhan_loop);  //<vs_dtt2>
    if(t.tdensity==DDT_DDT2Full)Interaction_ForcesGpuT_NN_gt3<tker,ftmode,tvisco,DDT_DDT2Full,shift>(t,time_inc,Zhan_loop);  //<vs_dtt2>
  }
#endif
}

//==============================================================================
template<TpKernel tker,TpFtMode ftmode> void Interaction_ForcesNN_gt1(const StInterParmsg &t,double time_inc, int Zhan_loop) {
  //GFCheck how to add fast compilation of laminar viscosity
#ifdef FAST_COMPILATION
  if(t.tvisco!=VISCO_LaminarSPS)throw "Extra viscosity options are disabled for FastCompilation...";
  Interaction_ForcesNN_gt2<tker,ftmode,VISCO_LaminarSPS>(t,time_inc,Zhan_loop);
#else
  if(t.tvisco==VISCO_ConstEq)		      Interaction_ForcesNN_gt2<tker,ftmode,VISCO_ConstEq>(t,time_inc,Zhan_loop);
  else if(t.tvisco==VISCO_LaminarSPS)	Interaction_ForcesNN_gt2<tker,ftmode,VISCO_LaminarSPS>(t,time_inc,Zhan_loop);
  else if(t.tvisco==VISCO_Artificial)	Interaction_ForcesNN_gt2<tker,ftmode,VISCO_Artificial>(t,time_inc,Zhan_loop);
  else if(t.tvisco==VISCO_Hypoplasticity)Interaction_ForcesNN_gt2<tker,ftmode,VISCO_Hypoplasticity>(t,time_inc,Zhan_loop);
  else if(t.tvisco==VISCO_Elasticity)Interaction_ForcesNN_gt2<tker,ftmode,VISCO_Elasticity>(t,time_inc,Zhan_loop);
#endif
}
//==============================================================================

template<TpKernel tker> void Interaction_ForcesNN_gt0(const StInterParmsg &t,double time_inc, int Zhan_loop) {
#ifdef FAST_COMPILATION
  if(t.ftmode!=FTMODE_None)throw "Extra FtMode options are disabled for FastCompilation...";
  Interaction_ForcesNN_gt1<tker,FTMODE_None>(t,time_inc,Zhan_loop);
#else
  if(t.ftmode==FTMODE_None)    Interaction_ForcesNN_gt1<tker,FTMODE_None>(t,time_inc,Zhan_loop);
  else if(t.ftmode==FTMODE_Sph)Interaction_ForcesNN_gt1<tker,FTMODE_Sph>(t,time_inc,Zhan_loop);
  else if(t.ftmode==FTMODE_Ext)Interaction_ForcesNN_gt1<tker,FTMODE_Ext>(t,time_inc,Zhan_loop);
#endif 
} 

//==============================================================================
void Interaction_ForcesNN(const StInterParmsg &t,double time_inc,int Zhan_loop) {
#ifdef FAST_COMPILATION
  if(t.tkernel!=KERNEL_Wendland)throw "Extra kernels are disabled for FastCompilation...";
  Interaction_ForcesNN_gt0<KERNEL_Wendland>(t,time_inc,Zhan_loop);
#else
  if(t.tkernel==KERNEL_Wendland)     Interaction_ForcesNN_gt0<KERNEL_Wendland>(t,time_inc,Zhan_loop);
#ifndef DISABLE_KERNELS_EXTRA
  else if(t.tkernel==KERNEL_Cubic)   Interaction_ForcesNN_gt0<KERNEL_Cubic   >(t,time_inc,Zhan_loop);
#endif
#endif
}

//======================End of NN Templates=======================================

void ComputePress_NN(unsigned np, unsigned npb,tsymatrix3f *SpsTaug,float *pressg){
const unsigned npf=np-npb;
  if(npf){
    dim3 sgridf=GetSimpleGridSize(npf,SPHBSIZE);
    KerComputePress_NN <<<sgridf,SPHBSIZE>>> (np,npb,(float2*)SpsTaug,pressg);
  }
}
}//end of file